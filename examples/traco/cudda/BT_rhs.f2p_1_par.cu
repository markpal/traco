#include "hip/hip_runtime.h"
int k;
int j;
int i;
#include <omp.h>
#include<omp.h>
#include<stdlib.h>
#include<stdio.h>

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))

/**********************************************************/
/* Maximum width of linear memory bound to texture memory */
/**********************************************************/
/* width in bytes */
#define LMAX_WIDTH    134217728
/* width in words */
#define LMAX_WWIDTH  33554432
/**********************************/
/* Maximum memory pitch (in bytes)*/
/**********************************/
#define MAX_PITCH   262144
/****************************************/
/* Maximum allowed GPU global memory    */
/* (should be less than actual size ) */
/****************************************/
#define MAX_GMSIZE  1600000000
/****************************************/
/* Maximum allowed GPU shared memory    */
/****************************************/
#define MAX_SMSIZE  16384
/********************************************/
/* Maximum size of each dimension of a grid */
/********************************************/
#define MAX_GDIMENSION  65535
#define MAX_NDIMENSION  10000

#define BLOCK_SIZE  128


static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



int N = 50;
int N1;
int N2;
int N3;
int N4;
int DIM;
int DIM1;
int DIM2;
int DIM3;
__global__ void seq_kernel0(int * N1, int * N2, int * N3, float qs[DIM1][DIM2][DIM3], float rho_i[DIM1][DIM2][DIM3], float square[DIM1][DIM2][DIM3], float u[6][DIM1][DIM2][DIM3], float us[DIM1][DIM2][DIM3], float vs[DIM1][DIM2][DIM3], float ws[DIM1][DIM2][DIM3])
{
int i;
int j;
int k;
int rho_inv;
int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
int _gtid = (threadIdx.x+(_bid*blockDim.x));
k=_gtid;
if (k<=( * N1))
{
for (j=0; j<=( * N2); j ++ )
{
for (i=0; i<=( * N3); i ++ )
{
rho_inv=(1.0/u[1][i][j][k]);
rho_i[i][j][k]=rho_inv;
us[i][j][k]=(u[2][i][j][k]*rho_inv);
vs[i][j][k]=(u[3][i][j][k]*rho_inv);
ws[i][j][k]=(u[4][i][j][k]*rho_inv);
square[i][j][k]=((0.5*(((u[2][i][j][k]*u[2][i][j][k])+(u[3][i][j][k]*u[3][i][j][k]))+(u[4][i][j][k]*u[4][i][j][k])))*rho_inv);
qs[i][j][k]=(square[i][j][k]*rho_inv);
}
}
}
}

void seq(float u[6][DIM1][DIM2][DIM3], float us[DIM1][DIM2][DIM3], float vs[DIM1][DIM2][DIM3], float ws[DIM1][DIM2][DIM3], float qs[DIM1][DIM2][DIM3], float square[DIM1][DIM2][DIM3], float rho_i[DIM1][DIM2][DIM3])
{
int * gpu__N1;
int * gpu__N2;
int * gpu__N3;
float * gpu__qs;
float * gpu__rho_i;
float * gpu__square;
float * gpu__u;
float * gpu__us;
float * gpu__vs;
float * gpu__ws;
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N1)), gpuBytes));
dim3 dimBlock0(gpuNumThreads, 1, 1);
gpuNumBlocks=((int)ceil((((float)(1+N1))/128.0F)));
if ((gpuNumBlocks>MAX_GDIMENSION))
{
gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
gpuNumBlocks1=MAX_NDIMENSION;
}
else
{
gpuNumBlocks2=1;
gpuNumBlocks1=gpuNumBlocks;
}
dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
totalNumThreads=(gpuNumBlocks*gpuNumThreads);
CUDA_SAFE_CALL(hipMemcpy(gpu__N1, ( & N1), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N2)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N2, ( & N2), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N3)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N3, ( & N3), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__qs)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__qs, qs, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__rho_i)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__rho_i, rho_i, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__square)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__square, square, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__u)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__u, u, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__us)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__us, us, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__vs)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__vs, vs, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__ws)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__ws, ws, gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(N1, N2, N3, qs, rho_i, square, u, us, vs, ws) private(i, j, k, rho_inv)
#pragma cuda ainfo kernelid(0) procname(seq) 
#pragma cuda gpurun nog2cmemtr(N1, N2, N3, u) 
seq_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__N1, gpu__N2, gpu__N3, ((float (*)[DIM2][DIM3])gpu__qs), ((float (*)[DIM2][DIM3])gpu__rho_i), ((float (*)[DIM2][DIM3])gpu__square), ((float (*)[DIM1][DIM2][DIM3])gpu__u), ((float (*)[DIM2][DIM3])gpu__us), ((float (*)[DIM2][DIM3])gpu__vs), ((float (*)[DIM2][DIM3])gpu__ws));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(ws, gpu__ws, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__ws));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(vs, gpu__vs, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__vs));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(us, gpu__us, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__us));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__u));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(square, gpu__square, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__square));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(rho_i, gpu__rho_i, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__rho_i));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(qs, gpu__qs, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__qs));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N3));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N2));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N1));
return ;
}

int main(int argc, char * argv[])
{
/* number of processors */
int num_proc = 1;
float (* u)[6][DIM2][DIM3];
float (* square)[DIM2][DIM3];
float (* qs)[DIM2][DIM3];
float (* ws)[DIM2][DIM3];
float (* vs)[DIM2][DIM3];
float (* us)[DIM2][DIM3];
float (* rho_i)[DIM2][DIM3];
struct timeval s1;
struct timeval f1;
double duration;
int _ret_val_0;

////////////////////////////////
// CUDA Device Initialization //
////////////////////////////////
int deviceCount;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
if (deviceCount == 0) {
	fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
	exit(EXIT_FAILURE);
}
int dev = 0;
hipDeviceProp_t deviceProp;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
CUDA_SAFE_CALL(hipSetDevice(dev));


num_proc=atoi(argv[1]);
if ((argc>2))
{
N=atoi(argv[2]);
}
N1=(N2=(N3=(N4=N)));
DIM=(DIM1=(DIM2=(DIM3=(N+2))));
us=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
vs=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
ws=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
square=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
qs=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
rho_i=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
u=((float (* )[6][DIM2][DIM3])malloc(((((DIM1*DIM2)*DIM3)*6)*sizeof (float))));
/* variables to meassure time */
printf("Ilosc procesorow: %i \n\n", num_proc);
omp_set_num_threads(num_proc);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
gettimeofday(( & s1), ((void * )0));
if ((num_proc==1))
{
seq(u, us, vs, ws, qs, square, rho_i);
}
else
{
comp(u, us, vs, ws, qs, square, rho_i);
}
gettimeofday(( & f1), ((void * )0));
/* ----------------------------------- */
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
duration=((((double)f1.tv_sec)+(((double)f1.tv_usec)/1000000))-(((double)s1.tv_sec)+(((double)s1.tv_usec)/1000000)));
printf("Czas: %2.3f seconds\n", duration);
printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
printf("====> GPU Block Size: 128 \n");
printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
printf("====> CPU-GPU Mem Transfer Opt Level: 2\n");
printf("====> Cuda Malloc Opt Level: 0\n");
printf("====> local array reduction variable configuration = 1\n");
fflush(stdout);
fflush(stderr);
return _ret_val_0;
}

