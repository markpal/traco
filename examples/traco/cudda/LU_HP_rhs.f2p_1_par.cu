#include "hip/hip_runtime.h"
int k;
int j;
int i;
int m;
#include <omp.h>
#include<omp.h>
#include<stdlib.h>
#include<stdio.h>

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))

/**********************************************************/
/* Maximum width of linear memory bound to texture memory */
/**********************************************************/
/* width in bytes */
#define LMAX_WIDTH    134217728
/* width in words */
#define LMAX_WWIDTH  33554432
/**********************************/
/* Maximum memory pitch (in bytes)*/
/**********************************/
#define MAX_PITCH   262144
/****************************************/
/* Maximum allowed GPU global memory    */
/* (should be less than actual size ) */
/****************************************/
#define MAX_GMSIZE  1600000000
/****************************************/
/* Maximum allowed GPU shared memory    */
/****************************************/
#define MAX_SMSIZE  16384
/********************************************/
/* Maximum size of each dimension of a grid */
/********************************************/
#define MAX_GDIMENSION  65535
#define MAX_NDIMENSION  10000

#define BLOCK_SIZE  128


static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



int N = 50;
int N1;
int N2;
int N3;
int N4;
int DIM;
int DIM1;
int DIM2;
int DIM3;
__global__ void seq_kernel0(int * N1, int * N2, int * N3, float frct[6][DIM1][DIM2][DIM3], float qs[DIM1][DIM2][DIM3], float rho_i[DIM1][DIM2][DIM3], float rsd[6][DIM1][DIM2][DIM3], float u[6][DIM1][DIM2][DIM3])
{
int i;
int j;
int k;
int m;
int tmp;
int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
int _gtid = (threadIdx.x+(_bid*blockDim.x));
k=(_gtid+1);
if (k<=( * N1))
{
for (j=1; j<=( * N2); j+=1)
{
for (i=1; i<=( * N3); i+=1)
{
for (m=1; m<=5; m+=1)
{
rsd[m][i][j][k]=( - frct[m][i][j][k]);
}
tmp=(1.0/u[1][i][j][k]);
rho_i[i][j][k]=tmp;
qs[i][j][k]=((0.5*(((u[2][i][j][k]*u[2][i][j][k])+(u[3][i][j][k]*u[3][i][j][k]))+(u[4][i][j][k]*u[4][i][j][k])))*tmp);
}
}
}
}

void seq(float rsd[6][DIM1][DIM2][DIM3], float u[6][DIM1][DIM2][DIM3], float rho_i[DIM1][DIM2][DIM3], float qs[DIM1][DIM2][DIM3], float frct[6][DIM1][DIM2][DIM3])
{
int * gpu__N1;
int * gpu__N2;
int * gpu__N3;
float * gpu__frct;
float * gpu__qs;
float * gpu__rho_i;
float * gpu__rsd;
float * gpu__u;
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N1)), gpuBytes));
dim3 dimBlock0(gpuNumThreads, 1, 1);
gpuNumBlocks=((int)ceil((((float)N1)/128.0F)));
if ((gpuNumBlocks>MAX_GDIMENSION))
{
gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
gpuNumBlocks1=MAX_NDIMENSION;
}
else
{
gpuNumBlocks2=1;
gpuNumBlocks1=gpuNumBlocks;
}
dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
totalNumThreads=(gpuNumBlocks*gpuNumThreads);
CUDA_SAFE_CALL(hipMemcpy(gpu__N1, ( & N1), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N2)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N2, ( & N2), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N3)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N3, ( & N3), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__frct)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__frct, frct, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__qs)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__qs, qs, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__rho_i)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__rho_i, rho_i, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__rsd)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__rsd, rsd, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__u)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__u, u, gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(N1, N2, N3, frct, qs, rho_i, rsd, u) private(i, j, k, m, tmp)
#pragma cuda ainfo kernelid(0) procname(seq) 
#pragma cuda gpurun nog2cmemtr(N1, N2, N3, frct, u) 
seq_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__N1, gpu__N2, gpu__N3, ((float (*)[DIM1][DIM2][DIM3])gpu__frct), ((float (*)[DIM2][DIM3])gpu__qs), ((float (*)[DIM2][DIM3])gpu__rho_i), ((float (*)[DIM1][DIM2][DIM3])gpu__rsd), ((float (*)[DIM1][DIM2][DIM3])gpu__u));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__u));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(rsd, gpu__rsd, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__rsd));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(rho_i, gpu__rho_i, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__rho_i));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(qs, gpu__qs, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__qs));
gpuBytes=((((6*DIM1)*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__frct));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N3));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N2));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N1));
return ;
}

int main(int argc, char * argv[])
{
/* number of processors */
int num_proc = 1;
float (* frct)[6][DIM2][DIM3];
float (* rsd)[6][DIM2][DIM3];
float (* u)[6][DIM2][DIM3];
float (* qs)[DIM2][DIM3];
float (* rho_i)[DIM2][DIM3];
struct timeval s1;
struct timeval f1;
double duration;
int _ret_val_0;

////////////////////////////////
// CUDA Device Initialization //
////////////////////////////////
int deviceCount;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
if (deviceCount == 0) {
	fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
	exit(EXIT_FAILURE);
}
int dev = 0;
hipDeviceProp_t deviceProp;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
CUDA_SAFE_CALL(hipSetDevice(dev));


num_proc=atoi(argv[1]);
if ((argc>2))
{
N=atoi(argv[2]);
}
N1=(N2=(N3=N));
DIM=(DIM1=(DIM2=(DIM3=(N+2))));
rho_i=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
qs=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
u=((float (* )[DIM1][DIM2][DIM3])malloc(((((DIM1*DIM2)*DIM3)*6)*sizeof (float))));
rsd=((float (* )[DIM1][DIM2][DIM3])malloc(((((DIM1*DIM2)*DIM3)*6)*sizeof (float))));
frct=((float (* )[DIM1][DIM2][DIM3])malloc(((((DIM1*DIM2)*DIM3)*6)*sizeof (float))));
/* variables to meassure time */
printf("Ilosc procesorow: %i \n\n", num_proc);
omp_set_num_threads(num_proc);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
/* PETLA SEKWENCYJNA */
gettimeofday(( & s1), ((void * )0));
if ((num_proc==1))
{
seq(rsd, u, rho_i, qs, frct);
}
gettimeofday(( & f1), ((void * )0));
/* ----------------------------------- */
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
duration=((((double)f1.tv_sec)+(((double)f1.tv_usec)/1000000))-(((double)s1.tv_sec)+(((double)s1.tv_usec)/1000000)));
printf("Czas: %2.3f seconds\n", duration);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
printf("====> GPU Block Size: 128 \n");
printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
printf("====> CPU-GPU Mem Transfer Opt Level: 2\n");
printf("====> Cuda Malloc Opt Level: 0\n");
printf("====> local array reduction variable configuration = 1\n");
fflush(stdout);
fflush(stderr);
return _ret_val_0;
}

