#include "hip/hip_runtime.h"
int i;
int j;
int ip;
#include <omp.h>
#include<omp.h>
#include<stdlib.h>
#include<stdio.h>

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))

/**********************************************************/
/* Maximum width of linear memory bound to texture memory */
/**********************************************************/
/* width in bytes */
#define LMAX_WIDTH    134217728
/* width in words */
#define LMAX_WWIDTH  33554432
/**********************************/
/* Maximum memory pitch (in bytes)*/
/**********************************/
#define MAX_PITCH   262144
/****************************************/
/* Maximum allowed GPU global memory    */
/* (should be less than actual size ) */
/****************************************/
#define MAX_GMSIZE  1600000000
/****************************************/
/* Maximum allowed GPU shared memory    */
/****************************************/
#define MAX_SMSIZE  16384
/********************************************/
/* Maximum size of each dimension of a grid */
/********************************************/
#define MAX_GDIMENSION  65535
#define MAX_NDIMENSION  10000

#define BLOCK_SIZE  128


static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



int N = 200;
int N1;
int N2;
int N3;
int DIM;
int DIM1;
int DIM2;
__global__ void seq_kernel0(int * N1, int * N2, int * N3, float dxm1[DIM1][DIM2], float wdtdr[DIM1][DIM2], float * wxm1)
{
int i;
int ip;
int j;
int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
int _gtid = (threadIdx.x+(_bid*blockDim.x));
i=(_gtid+1);
if (i<=( * N1))
{
for (j=1; j<=( * N2); j ++ )
{
for (ip=1; ip<=( * N3); ip ++ )
{
wdtdr[i][j]=(wdtdr[i][j]+((wxm1[ip]*dxm1[ip][i])*dxm1[ip][j]));
}
}
}
}

void seq(float wdtdr[DIM1][DIM2], float wxm1[DIM1], float dxm1[DIM1][DIM2])
{
int * gpu__N1;
int * gpu__N2;
int * gpu__N3;
float * gpu__dxm1;
float * gpu__wdtdr;
float * gpu__wxm1;
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N1)), gpuBytes));
dim3 dimBlock0(gpuNumThreads, 1, 1);
gpuNumBlocks=((int)ceil((((float)N1)/128.0F)));
if ((gpuNumBlocks>MAX_GDIMENSION))
{
gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
gpuNumBlocks1=MAX_NDIMENSION;
}
else
{
gpuNumBlocks2=1;
gpuNumBlocks1=gpuNumBlocks;
}
dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
totalNumThreads=(gpuNumBlocks*gpuNumThreads);
CUDA_SAFE_CALL(hipMemcpy(gpu__N1, ( & N1), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N2)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N2, ( & N2), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N3)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N3, ( & N3), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__dxm1)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__dxm1, dxm1, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__wdtdr)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__wdtdr, wdtdr, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(DIM1*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__wxm1)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__wxm1, wxm1, gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(N1, N2, N3, dxm1, wdtdr, wxm1) private(i, ip, j)
#pragma cuda ainfo kernelid(0) procname(seq) 
#pragma cuda gpurun nog2cmemtr(N1, N2, N3, dxm1, wxm1) 
seq_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__N1, gpu__N2, gpu__N3, ((float (*)[DIM2])gpu__dxm1), ((float (*)[DIM2])gpu__wdtdr), gpu__wxm1);
gpuBytes=(DIM1*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__wxm1));
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(wdtdr, gpu__wdtdr, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__wdtdr));
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__dxm1));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N3));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N2));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N1));
return ;
}

int main(int argc, char * argv[])
{
/* number of processors */
int num_proc = 1;
float (* dxm1)[DIM2];
float (* wxm1);
float (* wdtdr)[DIM2];
struct timeval s1;
struct timeval f1;
double duration;
int _ret_val_0;

////////////////////////////////
// CUDA Device Initialization //
////////////////////////////////
int deviceCount;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
if (deviceCount == 0) {
	fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
	exit(EXIT_FAILURE);
}
int dev = 0;
hipDeviceProp_t deviceProp;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
CUDA_SAFE_CALL(hipSetDevice(dev));


num_proc=atoi(argv[1]);
if ((argc>2))
{
N=atoi(argv[2]);
}
DIM=(N1=(N2=(N3=(DIM1=(DIM2=N)))));
wdtdr=((float (* )[DIM2])malloc(((DIM1*DIM2)*sizeof (float))));
wxm1=((float (* ))malloc((DIM1*sizeof (float))));
dxm1=((float (* )[DIM2])malloc(((DIM1*DIM2)*sizeof (float))));
/* variables to meassure time */
printf("Ilosc procesorow: %i \n\n", num_proc);
omp_set_num_threads(num_proc);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
gettimeofday(( & s1), ((void * )0));
if ((num_proc==1))
{
seq(wdtdr, wxm1, dxm1);
}
gettimeofday(( & f1), ((void * )0));
/* ----------------------------------- */
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
duration=((((double)f1.tv_sec)+(((double)f1.tv_usec)/1000000))-(((double)s1.tv_sec)+(((double)s1.tv_usec)/1000000)));
printf("Czas sekwencyjny: %2.3f seconds\n", duration);
printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
printf("====> GPU Block Size: 128 \n");
printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
printf("====> CPU-GPU Mem Transfer Opt Level: 2\n");
printf("====> Cuda Malloc Opt Level: 0\n");
printf("====> local array reduction variable configuration = 1\n");
fflush(stdout);
fflush(stderr);
return _ret_val_0;
}

