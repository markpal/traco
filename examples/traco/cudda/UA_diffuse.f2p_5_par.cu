#include "hip/hip_runtime.h"
int t2;
int t3;
int t4;
int t1;
#include <omp.h>
#include<omp.h>
#include<stdlib.h>
#include<stdio.h>

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))

/**********************************************************/
/* Maximum width of linear memory bound to texture memory */
/**********************************************************/
/* width in bytes */
#define LMAX_WIDTH    134217728
/* width in words */
#define LMAX_WWIDTH  33554432
/**********************************/
/* Maximum memory pitch (in bytes)*/
/**********************************/
#define MAX_PITCH   262144
/****************************************/
/* Maximum allowed GPU global memory    */
/* (should be less than actual size ) */
/****************************************/
#define MAX_GMSIZE  1600000000
/****************************************/
/* Maximum allowed GPU shared memory    */
/****************************************/
#define MAX_SMSIZE  16384
/********************************************/
/* Maximum size of each dimension of a grid */
/********************************************/
#define MAX_GDIMENSION  65535
#define MAX_NDIMENSION  10000

#define BLOCK_SIZE  128


static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



int N = 50;
int N1;
int N2;
int N3;
int N4;
int DIM;
int DIM1;
int DIM2;
int DIM3;
__global__ void seq_kernel0(int * N1, int * N2, int * N3, int * N4, float r[DIM1][DIM2][DIM3], float u[DIM1][DIM2][DIM3], float wdtdr[DIM1][DIM2])
{
int t1;
int t2;
int t3;
int t4;
int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
int _gtid = (threadIdx.x+(_bid*blockDim.x));
t2=(_gtid+1);
if (t2<=( * N2))
{
for (t3=1; t3<=( * N3); t3 ++ )
{
for (t4=1; t4<=( * N4); t4 ++ )
{
r[t4][t3][t2]=(r[t4][t3][t2]+(u[t4][t3][1]*wdtdr[1][t2]));
if (((((((t2<=( * N2))&&(t4>=1))&&(t4<=( * N4)))&&(t3>=1))&&(( * N3)>=t3))&&(t2>=1)))
{
for (t1=2; t1<=( * N1); t1 ++ )
{
r[t4][t3][t2]=(r[t4][t3][t2]+(u[t4][t3][t1]*wdtdr[t1][t2]));
}
}
}
}
}
}

void seq(float u[DIM1][DIM2][DIM3], float wdtdr[DIM1][DIM2], float r[DIM1][DIM2][DIM3])
{
int * gpu__N1;
int * gpu__N2;
int * gpu__N3;
int * gpu__N4;
float * gpu__r;
float * gpu__u;
float * gpu__wdtdr;
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N1)), gpuBytes));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N2)), gpuBytes));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N3)), gpuBytes));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N4)), gpuBytes));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__r)), gpuBytes));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__u)), gpuBytes));
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__wdtdr)), gpuBytes));
if ((((N3>=1)&&(N1>=2))&&(N4>=1)))
{
dim3 dimBlock0(gpuNumThreads, 1, 1);
gpuNumBlocks=((int)ceil((((float)N2)/128.0F)));
if ((gpuNumBlocks>MAX_GDIMENSION))
{
gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
gpuNumBlocks1=MAX_NDIMENSION;
}
else
{
gpuNumBlocks2=1;
gpuNumBlocks1=gpuNumBlocks;
}
dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
totalNumThreads=(gpuNumBlocks*gpuNumThreads);
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__N1, ( & N1), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__N2, ( & N2), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__N3, ( & N3), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMemcpy(gpu__N4, ( & N4), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(gpu__r, r, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(gpu__u, u, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(gpu__wdtdr, wdtdr, gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(N1, N2, N3, N4, r, u, wdtdr) private(t1, t2, t3, t4)
#pragma cuda ainfo kernelid(0) procname(seq) 
#pragma cuda gpurun nog2cmemtr(N1, N2, N3, N4, u, wdtdr) 
seq_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__N1, gpu__N2, gpu__N3, gpu__N4, ((float (*)[DIM2][DIM3])gpu__r), ((float (*)[DIM2][DIM3])gpu__u), ((float (*)[DIM2])gpu__wdtdr));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(r, gpu__r, gpuBytes, hipMemcpyDeviceToHost));
}
gpuBytes=((DIM1*DIM2)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__wdtdr));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__u));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__r));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N4));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N3));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N2));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N1));
return ;
}

int main(int argc, char * argv[])
{
/* number of processors */
int num_proc = 1;
float (* u)[DIM2][DIM3];
float (* wdtdr)[DIM2];
float (* r)[DIM2][DIM3];
struct timeval s1;
struct timeval f1;
double duration;
int _ret_val_0;

////////////////////////////////
// CUDA Device Initialization //
////////////////////////////////
int deviceCount;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
if (deviceCount == 0) {
	fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
	exit(EXIT_FAILURE);
}
int dev = 0;
hipDeviceProp_t deviceProp;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
CUDA_SAFE_CALL(hipSetDevice(dev));


num_proc=atoi(argv[1]);
if ((argc>2))
{
N=atoi(argv[2]);
}
N1=(N2=(N3=(N4=N)));
DIM=(DIM1=(DIM2=(DIM3=(N+2))));
wdtdr=((float (* )[DIM2])malloc(((DIM1*DIM2)*sizeof (float))));
r=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
u=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
/* variables to meassure time */
printf("Ilosc procesorow: %i \n\n", num_proc);
omp_set_num_threads(num_proc);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
gettimeofday(( & s1), ((void * )0));
if ((num_proc==1))
{
seq(u, wdtdr, r);
}
gettimeofday(( & f1), ((void * )0));
/* ----------------------------------- */
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
duration=((((double)f1.tv_sec)+(((double)f1.tv_usec)/1000000))-(((double)s1.tv_sec)+(((double)s1.tv_usec)/1000000)));
printf("Czas sekwencyjny: %2.3f seconds\n", duration);
printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
printf("====> GPU Block Size: 128 \n");
printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
printf("====> CPU-GPU Mem Transfer Opt Level: 2\n");
printf("====> Cuda Malloc Opt Level: 0\n");
printf("====> local array reduction variable configuration = 1\n");
fflush(stdout);
fflush(stderr);
return _ret_val_0;
}

