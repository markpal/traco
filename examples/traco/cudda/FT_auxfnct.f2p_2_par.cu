#include "hip/hip_runtime.h"
int i;
int k;
int j;
#include <omp.h>
#include<omp.h>
#include<stdlib.h>
#include<stdio.h>

#ifndef __O2G_HEADER__ 

#define __O2G_HEADER__ 

/******************************************/
/* Added codes for OpenMP2GPU translation */
/******************************************/
#include <cutil.h>
#include <math.h>
#define MAX(a,b) (((a) > (b)) ? (a) : (b))

/**********************************************************/
/* Maximum width of linear memory bound to texture memory */
/**********************************************************/
/* width in bytes */
#define LMAX_WIDTH    134217728
/* width in words */
#define LMAX_WWIDTH  33554432
/**********************************/
/* Maximum memory pitch (in bytes)*/
/**********************************/
#define MAX_PITCH   262144
/****************************************/
/* Maximum allowed GPU global memory    */
/* (should be less than actual size ) */
/****************************************/
#define MAX_GMSIZE  1600000000
/****************************************/
/* Maximum allowed GPU shared memory    */
/****************************************/
#define MAX_SMSIZE  16384
/********************************************/
/* Maximum size of each dimension of a grid */
/********************************************/
#define MAX_GDIMENSION  65535
#define MAX_NDIMENSION  10000

#define BLOCK_SIZE  128


static int gpuNumThreads = BLOCK_SIZE;
static int gpuNumBlocks;
static int gpuNumBlocks1;
static int gpuNumBlocks2;
static int totalNumThreads;
unsigned int gpuGmemSize = 0;
unsigned int gpuSmemSize = 0;
static unsigned int gpuBytes = 0;

#endif 
/* End of __O2G_HEADER__ */



int N = 50;
int N1;
int N2;
int N3;
int N4;
int DIM;
int DIM1;
int DIM2;
int DIM3;
__global__ void seq_kernel0(int * N1, int * N2, int * N3, float twiddle[DIM1][DIM2][DIM3], float x[DIM1][DIM2][DIM3], float y[DIM1][DIM2][DIM3])
{
int i;
int j;
int k;
int _bid = (blockIdx.x+(blockIdx.y*gridDim.x));
int _gtid = (threadIdx.x+(_bid*blockDim.x));
i=(_gtid+1);
if (i<=( * N1))
{
for (k=1; k<=( * N2); k ++ )
{
for (j=1; j<=( * N3); j ++ )
{
y[j][k][i]=(y[j][k][i]*twiddle[j][k][i]);
x[j][k][i]=y[j][k][i];
}
}
}
}

void seq(float x[DIM1][DIM2][DIM3], float y[DIM1][DIM2][DIM3], float twiddle[DIM1][DIM2][DIM3])
{
int * gpu__N1;
int * gpu__N2;
int * gpu__N3;
float * gpu__twiddle;
float * gpu__x;
float * gpu__y;
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N1)), gpuBytes));
dim3 dimBlock0(gpuNumThreads, 1, 1);
gpuNumBlocks=((int)ceil((((float)N1)/128.0F)));
if ((gpuNumBlocks>MAX_GDIMENSION))
{
gpuNumBlocks2=((int)ceil((((float)gpuNumBlocks)/10000.0F)));
gpuNumBlocks1=MAX_NDIMENSION;
}
else
{
gpuNumBlocks2=1;
gpuNumBlocks1=gpuNumBlocks;
}
dim3 dimGrid0(gpuNumBlocks1, gpuNumBlocks2, 1);
gpuNumBlocks=(gpuNumBlocks1*gpuNumBlocks2);
totalNumThreads=(gpuNumBlocks*gpuNumThreads);
CUDA_SAFE_CALL(hipMemcpy(gpu__N1, ( & N1), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N2)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N2, ( & N2), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__N3)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__N3, ( & N3), gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__twiddle)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__twiddle, twiddle, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__x)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__x, x, gpuBytes, hipMemcpyHostToDevice));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMalloc(((void * *)( & gpu__y)), gpuBytes));
CUDA_SAFE_CALL(hipMemcpy(gpu__y, y, gpuBytes, hipMemcpyHostToDevice));
#pragma omp parallel for shared(N1, N2, N3, twiddle, x, y) private(i, j, k)
#pragma cuda ainfo kernelid(0) procname(seq) 
#pragma cuda gpurun nog2cmemtr(N1, N2, N3, twiddle) 
seq_kernel0<<<dimGrid0, dimBlock0, 0, 0>>>(gpu__N1, gpu__N2, gpu__N3, ((float (*)[DIM2][DIM3])gpu__twiddle), ((float (*)[DIM2][DIM3])gpu__x), ((float (*)[DIM2][DIM3])gpu__y));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(y, gpu__y, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__y));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipMemcpy(x, gpu__x, gpuBytes, hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipFree(gpu__x));
gpuBytes=(((DIM1*DIM2)*DIM3)*sizeof (float));
CUDA_SAFE_CALL(hipFree(gpu__twiddle));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N3));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N2));
gpuBytes=sizeof (int);
CUDA_SAFE_CALL(hipFree(gpu__N1));
return ;
}

int main(int argc, char * argv[])
{
/* number of processors */
int num_proc = 1;
float (* y)[DIM2][DIM3];
float (* x)[DIM2][DIM3];
float (* twiddle)[DIM2][DIM3];
struct timeval s1;
struct timeval f1;
double duration;
int _ret_val_0;

////////////////////////////////
// CUDA Device Initialization //
////////////////////////////////
int deviceCount;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
if (deviceCount == 0) {
	fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
	exit(EXIT_FAILURE);
}
int dev = 0;
hipDeviceProp_t deviceProp;
CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
	fprintf(stderr, "Using device %d: %s\n", dev, deviceProp.name);
CUDA_SAFE_CALL(hipSetDevice(dev));


num_proc=atoi(argv[1]);
if ((argc>2))
{
N=atoi(argv[2]);
}
N1=(N2=(N3=N));
DIM=(DIM1=(DIM2=(DIM3=(N+2))));
x=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
y=((float (* )[DIM2][DIM3])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
twiddle=((float (* )[DIM1][DIM2])malloc((((DIM1*DIM2)*DIM3)*sizeof (float))));
/* variables to meassure time */
printf("Ilosc procesorow: %i \n\n", num_proc);
omp_set_num_threads(num_proc);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
/* PETLA SEKWENCYJNA */
gettimeofday(( & s1), ((void * )0));
if ((num_proc==1))
{
seq(x, y, twiddle);
}
gettimeofday(( & f1), ((void * )0));
/* ----------------------------------- */
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
duration=((((double)f1.tv_sec)+(((double)f1.tv_usec)/1000000))-(((double)s1.tv_sec)+(((double)s1.tv_usec)/1000000)));
printf("Czas: %2.3f seconds\n", duration);
/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */
printf("/***********************/ \n/* Input Configuration */ \n/***********************/ \n");
printf("====> GPU Block Size: 128 \n");
printf("/**********************/ \n/* Used Optimizations */ \n/**********************/ \n");
printf("====> CPU-GPU Mem Transfer Opt Level: 2\n");
printf("====> Cuda Malloc Opt Level: 0\n");
printf("====> local array reduction variable configuration = 1\n");
fflush(stdout);
fflush(stderr);
return _ret_val_0;
}

