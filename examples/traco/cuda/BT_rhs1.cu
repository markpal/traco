#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#define N 256

const long int N1 = N;  // Number of elements in arrays
const long int N2 = N;  // Number of elements in arrays
const long int N3 = N;  // Number of elements in arrays

const int DIM_N = N + 2;


// Kernel that executes on the CUDA device
 __global__ void par(float (*rho_i)[DIM_N][DIM_N], float (*u)[6][DIM_N][DIM_N], float (*square)[DIM_N][DIM_N], float (*vs)[DIM_N][DIM_N], float (*us)[DIM_N][DIM_N], float (*ws)[DIM_N][DIM_N], float (*qs)[DIM_N][DIM_N], int paczka)
{
      int idx = blockIdx.x;
      int t1,t2,t3;
      int lb = idx*paczka+1;
      int ub =  ((idx+1)*paczka < N1) ? (idx+1)*paczka : N1;
      float rho_inv;

   // lb--; ub--;
   if(lb==1) lb = 0;

    if (N2 >= 0 && N3 >= 0) {
      for(t1 = lb; t1 <= ub; t1++) {
        for(t2 = 0; t2 <= N2; t2++) {
          for(t3 = 0; t3 <= N3; t3++) {
            rho_inv = 1.0/u[1][t3][t2][t1];
         //   if (t1 >= 0 && t3 >= 0 && t2 >= 0 && N3 >= t3 && N1 >= t1 && N2 >= t2) {
              rho_i[t3][t2][t1] = rho_inv;
          //  }
          //  if (t1 >= 0 && t3 >= 0 && t2 >= 0 && N3 >= t3 && N1 >= t1 && N2 >= t2) {
              us[t3][t2][t1] = u[2][t3][t2][t1] * rho_inv;
          //  }
          //  if (t1 >= 0 && t3 >= 0 && t2 >= 0 && N3 >= t3 && N1 >= t1 && N2 >= t2) {
              vs[t3][t2][t1] = u[3][t3][t2][t1] * rho_inv;
          //  }
         //   if (t1 >= 0 && t3 >= 0 && t2 >= 0 && N3 >= t3 && N1 >= t1 && N2 >= t2) {
               ws[t3][t2][t1] = u[4][t3][t2][t1] * rho_inv;
         //   }
         //   if (t1 >= 0 && t3 >= 0 && t2 >= 0 && N3 >= t3 && N1 >= t1 && N2 >= t2) {
               square[t3][t2][t1] = 0.5* ( u[2][t3][t2][t1]*u[2][t3][t2][t1] + u[3][t3][t2][t1]*u[3][t3][t2][t1] + u[4][t3][t2][t1]*u[4][t3][t2][t1] ) * rho_inv;

          //  }
       //     if (t1 >= 0 && t3 >= 0 && t2 >= 0 && N3 >= t3 && N1 >= t1 && N2 >= t2) {
              qs[t3][t2][t1] = square[t3][t2][t1] * rho_inv;
         //   }
          }

          }
        }
      }

}



__global__ void seq(float (*rho_i)[DIM_N][DIM_N], float (*u)[6][DIM_N][DIM_N], float (*square)[DIM_N][DIM_N], float (*vs)[DIM_N][DIM_N], float (*us)[DIM_N][DIM_N], float (*ws)[DIM_N][DIM_N], float (*qs)[DIM_N][DIM_N])
{
	int k, i, j;
	float rho_inv;


    for(k = 0; k <= N1; k++){
      for(j = 0; j <= N2; j++){
        for(i = 0; i <= N3; i++){
          rho_inv = 1.0/u[1][i][j][k];
          rho_i[i][j][k] = rho_inv;
          us[i][j][k] = u[2][i][j][k] * rho_inv;
          vs[i][j][k] = u[3][i][j][k] * rho_inv;
          ws[i][j][k] = u[4][i][j][k] * rho_inv;
          square[i][j][k]     = 0.5* ( u[2][i][j][k]*u[2][i][j][k] + u[3][i][j][k]*u[3][i][j][k] + u[4][i][j][k]*u[4][i][j][k] ) * rho_inv;
          qs[i][j][k] = square[i][j][k] * rho_inv;
        }
      }
    }

}



int main(int argc, char **argv)
{
    float *rho_i_h, *rho_i_d;  // Pointer to host & device arrays
    float *rho_i_hs, *rho_i_ds;  // Pointer to host & device arrays

    float *u_h, *u_d;  // Pointer to host & device arrays

    float *us_h, *us_d;  // Pointer to host & device arrays
    float *us_hs, *us_ds;  // Pointer to host & device arrays

    float *vs_h, *vs_d;  // Pointer to host & device arrays
    float *vs_hs, *vs_ds;  // Pointer to host & device arrays

    float *ws_h, *ws_d;  // Pointer to host & device arrays
    float *ws_hs, *ws_ds;  // Pointer to host & device arrays

    float *qs_h, *qs_d;  // Pointer to host & device arrays
    float *qs_hs, *qs_ds;  // Pointer to host & device arrays

    float *square_h, *square_d;  // Pointer to host & device arrays
    float *square_hs, *square_ds;  // Pointer to host & device arrays


    struct timeval s1, f1;
	double  duration;

  	size_t size =  DIM_N * DIM_N * DIM_N * sizeof(float);
	int _size = DIM_N * DIM_N * DIM_N;

  	rho_i_h = (float *)malloc(size);        // Allocate array on host
  	rho_i_hs = (float *)malloc(size);        // Allocate array on host
  	us_h = (float *)malloc(size);
  	us_hs = (float *)malloc(size);
  	vs_h = (float *)malloc(size);
  	vs_hs = (float *)malloc(size);
  	ws_h = (float *)malloc(size);
  	ws_hs = (float *)malloc(size);
  	qs_h = (float *)malloc(size);
  	qs_hs = (float *)malloc(size);
  	square_h = (float *)malloc(size);
  	square_hs = (float *)malloc(size);

  	u_h = (float *)malloc(size*6);

	// Initialize host array
  	for (int i=0; i<_size*6; i++) u_h[i] = (float)i;
  	for (int i=0; i<_size; i++) qs_h[i] = (float)i;
  	for (int i=0; i<_size; i++) qs_hs[i] = (float)i;

	//  hipMalloc((void **) &pdiff_ds, size);   // Allocate array on device
	hipMalloc((void **) &rho_i_ds, size);   // Allocate array on device
	hipMalloc((void **) &us_ds, size);   // Allocate array on device
    hipMalloc((void **) &vs_ds, size);   // Allocate array on device
    hipMalloc((void **) &ws_ds, size);   // Allocate array on device
    hipMalloc((void **) &qs_ds, size);   // Allocate array on device
    hipMalloc((void **) &square_ds, size);   // Allocate array on device

    //hipMemcpy(rms_ds, rms_hs, sizeof(float)*6, hipMemcpyHostToDevice);
    hipMemcpy(qs_ds, qs_hs, size, hipMemcpyHostToDevice);


	//  copy it to CUDA device
	gettimeofday(&s1, NULL);
	hipMalloc((void **) &rho_i_d, size);   // Allocate array on device
	hipMalloc((void **) &us_d, size);   // Allocate array on device
    hipMalloc((void **) &vs_d, size);   // Allocate array on device
    hipMalloc((void **) &ws_d, size);   // Allocate array on device
    hipMalloc((void **) &qs_d, size);   // Allocate array on device
    hipMalloc((void **) &square_d, size);   // Allocate array on device
    hipMalloc((void **) &u_d, 6*size);   // Allocate array on device
    gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas alloc: %2.6f seconds", duration);

    hipMemcpy(qs_d, qs_h, size, hipMemcpyHostToDevice);
    gettimeofday(&s1, NULL);
  	hipMemcpy(u_d, u_h, size*6, hipMemcpyHostToDevice);

	//hipMemcpy(rhs_d, rhs_h, size, hipMemcpyHostToDevice);

    gettimeofday(&f1, NULL);
  	 // Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("\nCzas send: %2.6f seconds\n", duration);

      // SEQ --------------------
    gettimeofday(&s1, NULL);
//    seq<<<1, 1>>>((float(*)[DIM_N][DIM_N])rho_i_ds, (float(*)[6][DIM_N][DIM_N])u_d, (float(*)[DIM_N][DIM_N])square_ds, (float(*)[DIM_N][DIM_N])vs_ds, (float(*)[DIM_N][DIM_N])us_ds, (float(*)[DIM_N][DIM_N])ws_ds, (float(*)[DIM_N][DIM_N])qs_ds);
    hipDeviceSynchronize();
    gettimeofday(&f1, NULL);

	// hipMemcpy(pdiff_hs, pdiff_ds, sizeof(int)*_size, hipMemcpyDeviceToHost);
	hipMemcpy(rho_i_hs, rho_i_ds, size, hipMemcpyDeviceToHost);
	hipMemcpy(square_hs, square_ds, size, hipMemcpyDeviceToHost);
	hipMemcpy(vs_hs, vs_ds, size, hipMemcpyDeviceToHost);
	hipMemcpy(us_hs, us_ds, size, hipMemcpyDeviceToHost);
	hipMemcpy(ws_hs, ws_ds, size, hipMemcpyDeviceToHost);
	hipMemcpy(qs_hs, qs_ds, size, hipMemcpyDeviceToHost);


	// printf("%i %2.6f %2.6f\n", 0, tm2_hs[0], tm2_h[0]);
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas sekw.: %2.6f seconds\n", duration);

	//------------------------

	// PAR --------------------

    int block_size = 1;
  	int n_blocks = atoi(argv[1]);  // CPU
	int paczka = (int)ceil(N1 / n_blocks);
printf("%i", paczka);

    gettimeofday(&s1, NULL);
    par<<< n_blocks, block_size >>> ((float(*)[DIM_N][DIM_N])rho_i_d, (float(*)[6][DIM_N][DIM_N])u_d, (float(*)[DIM_N][DIM_N])square_d, (float(*)[DIM_N][DIM_N])vs_d, (float(*)[DIM_N][DIM_N])us_d, (float(*)[DIM_N][DIM_N])ws_d, (float(*)[DIM_N][DIM_N])qs_d, paczka);
    hipDeviceSynchronize();
    gettimeofday(&f1, NULL);


	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas par: %2.6f seconds\n", duration);

    //------------------------

	//------------------------

  	// Retrieve result from device and store it in host array
    gettimeofday(&s1, NULL);
    //hipMemcpy(x_h, x_d, sizeof(int)*size, hipMemcpyDeviceToHost);
	//hipMemcpy(pdiff_h, pdiff_d, sizeof(int)*_size, hipMemcpyDeviceToHost);
	hipMemcpy(rho_i_h, rho_i_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(square_h, square_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(vs_h, vs_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(us_h, us_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(ws_h, ws_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(qs_h, qs_d, size, hipMemcpyDeviceToHost);


	//hipMemcpy(wdtdr_h, wdtdr_d, sizeof(int)*N, hipMemcpyDeviceToHost);
	gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas fetch: %2.6f seconds\n", duration);


	for (int i=0; i<_size ; i++)
          if (qs_hs[i] != qs_h[i])
          {
            printf("%i %f %f\n", i, qs_hs[i], qs_h[i]);
            printf("blad\n");
            exit(0);
          }

    return 0;
}





