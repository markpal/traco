#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#include "common.h"

// Kernel that executes on the CUDA device
 __global__ void par1(float (*sum_tang)[MAXGRID][MAXGRID], float (*diff)[MAXGRID][MAXGRID][LENGTH], int MAXGRID, int LENGTH,  int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < MAXGRID-1) ? (idx+1)*paczka : MAXGRID-1;
      int i,j,cnt;
	  float sum;

      for (j=lb; j<=ub; j++) {
			for (i=0; i<=MAXGRID-1; i++) {
				sum = sum_tang[j][i];
				for (cnt=0; cnt<=LENGTH-1; cnt++) {
					diff[j][i][cnt] = sum;
				}
			}
       }


}


 __global__ void par2(float (*mean)[MAXGRID][MAXGRID], float (*sum_diff)[MAXGRID][MAXGRID][LENGTH], float (*diff)[MAXGRID][MAXGRID][LENGTH], int MAXGRID, int LENGTH,  int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < MAXGRID-1) ? (idx+1)*paczka : MAXGRID-1;
      int i,j,cnt;
	  float sum;

      for (j=lb; j<=ub; j++) {
		for (i=j; i<=MAXGRID-1; i++) {
			sum_diff[j][i][0] = diff[j][i][0];
			for (cnt=1; cnt<=LENGTH-1; cnt++) {
                sum_diff[j][i][cnt] = sum_diff[j][i][cnt-1] + diff[j][i][cnt];
            }
			mean[j][i] = sum_diff[j][i][LENGTH-1];
        }
      }
}


 __global__ void par3(float (*path)[MAXGRID][MAXGRID], float (*mean)[MAXGRID][MAXGRID], int MAXGRID, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < MAXGRID-1) ? (idx+1)*paczka : MAXGRID-1;
      int i,j,cnt;
	  float sum;


      for (i=lb; i<=ub; i++) {
			path[0][i] = mean[0][i];
      }

      __synchtreads();

      for (j=lb; j<=ub; j++) {
			for (i=j; i<=MAXGRID-1; i++) {
				path[j][i] = path[j-1][i-1] + mean[j][i];
			}
       }
}




 __global__ void seq(float (*sum_tang)[MAXGRID][MAXGRID], float (*sum_diff)[MAXGRID][MAXGRID][LENGTH], float (*diff)[MAXGRID][MAXGRID][LENGTH], float (*path)[MAXGRID][MAXGRID], float (*mean)[MAXGRID][MAXGRID], int MAXGRID, int LENGTH, int ITER, int paczka)
{

	for (t=0; t<ITER; t++) {
		for (j=0; j<=MAXGRID-1; j++) {
			for (i=0; i<=MAXGRID-1; i++) {
				sum = sum_tang[j][i];
				for (cnt=0; cnt<=LENGTH-1; cnt++) {
					diff[j][i][cnt] = sum;
				}
			}
		}
		for (j=0; j<=MAXGRID-1; j++) {
			for (i=j; i<=MAXGRID-1; i++) {
				sum_diff[j][i][0] = diff[j][i][0];
				for (cnt=1; cnt<=LENGTH-1; cnt++) {
					sum_diff[j][i][cnt] = sum_diff[j][i][cnt-1] + diff[j][i][cnt];
				}
				mean[j][i] = sum_diff[j][i][LENGTH-1];
			}
		}

		for (i=0; i<=MAXGRID-1; i++) {
			path[0][i] = mean[0][i];
		}

		for (j=1; j<=MAXGRID-1; j++) {
			for (i=j; i<=MAXGRID-1; i++) {
				path[j][i] = path[j-1][i-1] + mean[j][i];
			}
		}
	}
}

  	// main routine that executes on the host
int main(int argc, char **argv)
{
  	  float *sum_tang_h, *sum_tang_d;  // Pointer to host & device arrays
  	  float *sum_diff_h, *sum_diff_d;  // Pointer to host & device arrays
  	  float *sum_diff_hs, *sum_diff_ds;  // Pointer to host & device arrays
      float *diff_h, *diff_d;  // Pointer to host & device arrays
  	  float *diff_hs, *diff_ds;  // Pointer to host & device arrays
  	  float *mean_h, *mean_d;  // Pointer to host & device arrays
   	  float *mean_hs, *mean_ds;  // Pointer to host & device arrays
   	  float *path_h, *path_d;  // Pointer to host & device arrays
   	  float *path_hs, *path_ds;  // Pointer to host & device arrays

	  struct timeval s1, f1;
	  double  duration;

  	  size_t size =  MAXGRID * MAXGRID * LENGTH * sizeof(float);
	  int _size = MAXGRID * MAXGRID * LENGTH;

  	  size_t size2 =  MAXGRID * LENGTH * sizeof(float);
	  int _size2 = MAXGRID * LENGTH;

  	  sum_tang_h = (float *)malloc(size2);        // Allocate array on host
	  sum_diff_h = (float *)malloc(size);        // Allocate array on host
  	  sum_diff_hs = (float *)malloc(size);        // Allocate array on host
  	  diff_h = (float *)malloc(size);        // Allocate array on host
  	  diff_hs = (float *)malloc(size);        // Allocate array on host
  	  mean_h = (float *)malloc(size2);        // Allocate array on host
  	  mean_hs = (float *)malloc(size2);        // Allocate array on host
      path_h = (float *)malloc(size2);        // Allocate array on host
  	  path_hs = (float *)malloc(size2);        // Allocate array on host

      // Set the input data
        for (i=0; i<MAXGRID; i++) {
            for (j=0; j<MAXGRID; j++) {
                sum_tang_h[i][j] = (float)((i+1)*(j+1));
                mean_h[i][j] = ((float) i-j) / MAXGRID;
                path_h[i][j] = ((float) i*(j-1)) / MAXGRID;
            }
        }

	  hipMalloc((void **) &sum_diff_ds, size);   // Allocate array on device
	  hipMalloc((void **) &diff_ds, size);   // Allocate array on device
	  hipMalloc((void **) &mean_ds, size2);   // Allocate array on device
	  hipMalloc((void **) &path_ds, size2);   // Allocate array on device

	  hipMemcpy(sum_diff_ds, sum_diff_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(diff_ds, diff_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(mean_ds, mean_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(path_ds, path_h, size, hipMemcpyHostToDevice);

	  //  copy it to CUDA device
	  gettimeofday(&s1, NULL);

	  hipMalloc((void **) &sum_diff_d, size);   // Allocate array on device
	  hipMalloc((void **) &sum_tang_d, size2);   // Allocate array on device
	  hipMalloc((void **) &diff_d, size);   // Allocate array on device
	  hipMalloc((void **) &mean_d, size2);   // Allocate array on device
	  hipMalloc((void **) &path_d, size2);   // Allocate array on device

      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas alloc: %2.6f seconds\n", duration);

	  gettimeofday(&s1, NULL);
  	  hipMemcpy(sum_tang_d, sum_tang_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(mean_d, mean_h, size2, hipMemcpyHostToDevice);
      hipMemcpy(path_d, path_h, size2, hipMemcpyHostToDevice);

      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas copy: %2.6f seconds\n", duration);


	  // SEQ --------------------
      gettimeofday(&s1, NULL);
      seq<<<1, 1>>>((float(*)[MAXGRID][MAXGRID])sum_tang_d, (float(*)[MAXGRID][MAXGRID][LENGTH])sum_diff_ds, (float(*)[MAXGRID][MAXGRID][LENGTH])diff_ds, (float(*)[MAXGRID][MAXGRID])path_ds,  (float(*)[MAXGRID][MAXGRID])mean_ds, MAXGRID, LENGTH, ITER, paczka)
      hipDeviceSynchronize();
      gettimeofday(&f1, NULL);

	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas sekw.: %2.6f seconds\n", duration);

  	  hipMemcpy(diff_hs, diff_ds, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(sum_diff_hs, sum_diff_ds, sizeof(float)*size, hipMemcpyDeviceToHost);
      hipMemcpy(mean_hs, mean_ds, sizeof(float)*size2, hipMemcpyDeviceToHost);
  	  hipMemcpy(mean_hs, mean_ds, sizeof(float)*size2, hipMemcpyDeviceToHost);
	  hipFree(diff_ds);
	  hipFree(sum_diff_ds);
	  hipFree(mean_ds);
	  hipFree(path_ds);

	  //------------------------

	  // PAR --------------------

      int block_size = 1;
  	  int n_blocks = atoi(argv[1]);  // CPU
	  int paczka = (int)ceil(N / n_blocks);
         // printf("\n*** %d \n", paczka);

      gettimeofday(&s1, NULL);
	  for (t=0; t<ITER; t++) {
        par1<<< n_blocks, block_size >>> ((float(*)[MAXGRID][MAXGRID])sum_tang_d, (float(*)[MAXGRID][MAXGRID][LENGTH])diff_d, MAXGRID, LENGTH, paczka);
        hipDeviceSynchronize();
        par2<<< n_blocks, block_size >>> ((float(*)[MAXGRID][MAXGRID])mean_d, (float(*)[MAXGRID][MAXGRID][LENGTH])sum_diff_d, (float(*)[MAXGRID][MAXGRID][LENGTH])diff_d, MAXGRID, LENGTH, paczka);
        hipDeviceSynchronize();
        par3<<< n_blocks, block_size >>> ((float(*)[MAXGRID][MAXGRID])path_d, (float(*)[MAXGRID][MAXGRID])mean_d, MAXGRID, paczka);
        hipDeviceSynchronize();
	  }
	  hipDeviceSynchronize();
      gettimeofday(&f1, NULL);


	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas par: %2.6f seconds\n", duration);

	  //------------------------

  	  // Retrieve result from device and store it in host array
      gettimeofday(&s1, NULL);
      //hipMemcpy(x_h, x_d, sizeof(float)*size, hipMemcpyDeviceToHost);
      hipMemcpy(diff_h, diff_d, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(sum_diff_h, sum_diff_d, sizeof(float)*size, hipMemcpyDeviceToHost);
      hipMemcpy(mean_h, mean_d, sizeof(float)*size2, hipMemcpyDeviceToHost);
  	  hipMemcpy(mean_h, mean_d, sizeof(float)*size2, hipMemcpyDeviceToHost);
	  //hipMemcpy(twiddle_h, twiddle_d, sizeof(float)*N, hipMemcpyDeviceToHost);
      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas fetch: %2.6f seconds\n", duration);

/*
	  for (int i=0; i<_size ; i++)
	  if (y_hs[i] != y_h[i])
          {
		printf("%i %2.6f %2.6f\n", i, y_hs[i], y_h[i]);
	        printf("blad\n");
		exit(0);
	  }
*/

  	  // Print results
  	  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  	  // Cleanup
      hipFree(diff_d);
      hipFree(sum_tang_d);
	  hipFree(sum_diff_d);
	  hipFree(mean_d);
	  hipFree(path_d);

      free(diff_h);
      free(sum_tang_h);
	  free(sum_diff_h);
	  free(mean_h);
	  free(path_h);

}
