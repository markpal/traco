#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#include "common.h"

// Kernel that executes on the CUDA device
 __global__ void par1(float (*A)[N][N], float (*B)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka+1;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i,j;


      for (i=lb; i<ub; i++) {
			for (j=1; j<N-1; j++) {
				B[i][j] = 0.2 * (A[i][j] + A[i][j-1] + A[i][1+j] + A[1+i][j] + A[i-1][j]);
			}
	   }


}

 __global__ void par2(float (*A)[N][N], float (*B)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka+1;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i,j;


	  for (i=lb; i<ub; i++) {
			for (j=1; j<N-1; j++) {
				A[i][j] = B[i][j];
			}
      }


}



 __global__ void seq(float (*A)[N][N], float (*B)[N][N], int N, int TSTEPS, int paczka)
{
    int i,j,t;

	for (t=0; t<TSTEPS; t++) {
		for (i=1; i<N-1; i++) {
			for (j=1; j<N-1; j++) {
				B[i][j] = 0.2 * (A[i][j] + A[i][j-1] + A[i][1+j] + A[1+i][j] + A[i-1][j]);
			}
		}
		for (i=1; i<N-1; i++) {
			for (j=1; j<N-1; j++) {
				A[i][j] = B[i][j];
			}
		}
	}
}

  	// main routine that executes on the host
int main(int argc, char **argv)
{
  	  int t;

  	  float *A_h, *A_d;  // Pointer to host & device arrays
  	  float *B_h, *B_d;  // Pointer to host & device arrays
  	  float *A_hs, *A_ds;  // Pointer to host & device arrays
  	  float *B_hs, *B_ds;  // Pointer to host & device arrays

	  struct timeval s1, f1;
	  double  duration;

  	  size_t size =  N * N * sizeof(float);
	  int _size = N * N * LENGTH;


  	  A_h = (float *)malloc(size);        // Allocate array on host
  	  A_hs = (float *)malloc(size);        // Allocate array on host
  	  B_h = (float *)malloc(size);        // Allocate array on host
  	  B_hs = (float *)malloc(size);        // Allocate array on host

        // Set the input data
        for (i=0; i<N; i++) {
            for (j=0; j<N; j++) {
                A_h[i][j] = ((float) i*(j+2) + 2) / N;
                B_h[i][j] = ((float) i*(j+3) + 3) / N;
            }
        }

	  hipMalloc((void **) &A_ds, size);   // Allocate array on device
	  hipMalloc((void **) &B_ds, size);   // Allocate array on device

	  hipMemcpy(A_ds, A_h, size, hipMemcpyHostToDevice);

	  //  copy it to CUDA device
	  gettimeofday(&s1, NULL);

	  hipMalloc((void **) &A_d, size);   // Allocate array on device
	  hipMalloc((void **) &B_d, size);   // Allocate array on device

      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas alloc: %2.6f seconds\n", duration);

	  gettimeofday(&s1, NULL);
  	  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas copy: %2.6f seconds\n", duration);


	  // SEQ --------------------
      gettimeofday(&s1, NULL);
      seq<<<1, 1>>>((float(*)[N][N])A_ds, (float(*)[N][N]B_ds, N, TSTEPS, paczka)
      hipDeviceSynchronize();
      gettimeofday(&f1, NULL);

	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas sekw.: %2.6f seconds\n", duration);

  	  hipMemcpy(A_hs, A_ds, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(B_hs, B_ds, sizeof(float)*size, hipMemcpyDeviceToHost);

	  hipFree(A_ds);
	  hipFree(B_ds);


	  //------------------------

	  // PAR --------------------

      int block_size = 1;
  	  int n_blocks = atoi(argv[1]);  // CPU
	  int paczka = (int)ceil(N / n_blocks);
         // printf("\n*** %d \n", paczka);

      gettimeofday(&s1, NULL);
	  for (t=0; t<ITER; t++) {
        par1<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, N, paczka);
        hipDeviceSynchronize();
        par1<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, N, paczka);
        hipDeviceSynchronize();
	  }
	  hipDeviceSynchronize();
      gettimeofday(&f1, NULL);


	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas par: %2.6f seconds\n", duration);

	  //------------------------

  	  // Retrieve result from device and store it in host array
      gettimeofday(&s1, NULL);
      //hipMemcpy(x_h, x_d, sizeof(float)*size, hipMemcpyDeviceToHost);
      hipMemcpy(A_h, A_d, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(B_h, B_d, sizeof(float)*size, hipMemcpyDeviceToHost);
	  //hipMemcpy(twiddle_h, twiddle_d, sizeof(float)*N, hipMemcpyDeviceToHost);
      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas fetch: %2.6f seconds\n", duration);

/*
	  for (int i=0; i<_size ; i++)
	  if (y_hs[i] != y_h[i])
          {
		printf("%i %2.6f %2.6f\n", i, y_hs[i], y_h[i]);
	        printf("blad\n");
		exit(0);
	  }
*/

  	  // Print results
  	  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  	  // Cleanup
      hipFree(A_d);
      hipFree(B_d);

      free(A_h);
      free(B_h);


}
