#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#define N 512
const int N1 = N;
const int N2 = N;
const int N3 = N;

const int DIM_N = N + 2;


// Kernel that executes on the CUDA device
 __global__ void par(float (*tmp)[DIM_N], float (*tmor)[DIM_N], float (*qbnew)[DIM_N][2], int paczka)
{
  	  int idx = blockIdx.x;
      int t1, t2, t3, i;
      int lb = idx*paczka+1;
      int ub =  ((idx+1)*paczka < N1) ? (idx+1)*paczka : N1;


      if (N2 >= 2 && N3 >= 2) {
      for(t1 = lb; t1 <= ub; t1++){
        for(t2 = 2; t2 <= N2; t2++) {
          tmp[t2][t1] = tmp[t2][t1] + qbnew[t2-1][1][1]*tmor[1][t1];
          if (N2 >= t2 && N1 >= t1 && t2 >= 2 && t1 >= 1) {
            for(t3 = 2; t3 <= N3; t3++) {
              tmp[t2][t1] = tmp[t2][t1] + qbnew[t2-1][t3][1]*tmor[t3][t1];
            }
          }

        }
      }
    }

    for(t1 = lb; t1 <= ub; t1++) {
      i=1;
      if (t1 >= 1 && N1 >= t1) {
        tmp[i][t1]=tmor[i][t1];
     }

   }


}




 __global__ void seq(float (*tmp)[DIM_N], float (*tmor)[DIM_N], float (*qbnew)[DIM_N][2])
{
	int col, i, j;


    for(col = 1; col <= N1; col++){
      i = 1;
      tmp[i][col]=tmor[i][col];
      for(i = 2; i <= N2; i++){
        for(j = 1; j <= N3; j++){
          tmp[i][col] = tmp[i][col] + qbnew[i-1][j][1]*tmor[j][col];
        }
      }
    }


}






int main(int argc, char **argv)
{
  	float *tmp_h, *tmp_d;  // Pointer to host & device arrays
   	float *tmp_hs, *tmp_ds;  // Pointer to host & device arrays

    float *qbnew_h, *qbnew_d;  // Pointer to host & device arrays
    float *tmor_h, *tmor_d;  // Pointer to host & device arrays

	struct timeval s1, f1;
	double  duration;

  	size_t size =  DIM_N * DIM_N * sizeof(float);
	int _size = DIM_N * DIM_N ;

    tmp_h = (float *)malloc(size);        // Allocate array on host
	tmp_hs = (float *)malloc(size);        // Allocate array on host
  	tmor_h = (float *)malloc(size);        // Allocate array on host
  	qbnew_h = (float *)malloc(size*2);        // Allocate array on host

  	for (int i=0; i<_size; i++) tmp_h[i] = (float)i;
	for (int i=0; i<_size; i++) tmp_hs[i] = (float)i;
	for (int i=0; i<_size; i++) tmor_h[i] = (float)i;
	for (int i=0; i<_size; i++) qbnew_h[i] = (float)i;

	hipMalloc((void **) &tmp_ds, size);   // Allocate array on device

	hipMemcpy(tmp_ds, tmp_hs, size, hipMemcpyHostToDevice);

//  copy it to CUDA device
	gettimeofday(&s1, NULL);

	hipMalloc((void **) &tmp_d, size);
	hipMalloc((void **) &tmor_d, size);   // Allocate array on device
    hipMalloc((void **) &qbnew_d, size*2);   // Allocate array on device

    gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas alloc: %2.6f seconds", duration);

    gettimeofday(&s1, NULL);

  	hipMemcpy(tmp_d, tmp_h, size, hipMemcpyHostToDevice);
	hipMemcpy(tmor_d, tmor_h, size, hipMemcpyHostToDevice);
    hipMemcpy(qbnew_d, qbnew_h, size*2, hipMemcpyHostToDevice);

    gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas send: %2.6f seconds\n", duration);

	gettimeofday(&s1, NULL);
//    seq<<<1, 1>>>((float(*)[DIM_N])tmp_ds, (float(*)[DIM_N])tmor_d, (float(*)[DIM_N][2])qbnew_d);
    hipDeviceSynchronize();
    gettimeofday(&f1, NULL);

    hipMemcpy(tmp_hs, tmp_ds, size, hipMemcpyDeviceToHost);

    duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas sekw.: %2.6f seconds\n", duration);

	// PAR --------------------

    int block_size = 1;
  	int n_blocks = atoi(argv[1]);  // CPU
	int paczka = (int)ceil(N1 / n_blocks);


    gettimeofday(&s1, NULL);
	par<<< n_blocks, block_size >>> ((float(*)[DIM_N])tmp_d, (float(*)[DIM_N])tmor_d, (float(*)[DIM_N][2])qbnew_d, paczka);
    hipDeviceSynchronize();
    gettimeofday(&f1, NULL);

    duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas par: %2.6f seconds\n", duration);

	//------------------------
    // Retrieve result from device and store it in host array
    gettimeofday(&s1, NULL);
	hipMemcpy(tmp_h, tmp_d, size, hipMemcpyDeviceToHost);
    gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas fetch: %2.6f seconds\n", duration);

    for (int i=0; i<_size ; i++)
	  if (tmp_hs[i] != tmp_h[i])
          {
		printf("%i %f %f\n", i, tmp_hs[i], tmp_h[i]);
	        printf("blad\n");
		exit(0);
	  }

	  // Cleanup
  	  free(tmp_h); hipFree(tmp_d);
	  free(tmp_hs); hipFree(tmp_ds);
      free(tmor_h); hipFree(tmor_d);
	  free(qbnew_h); hipFree(qbnew_d);

    return 0;
}
