#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

const long int N = 1000;  // Number of elements in arrays
const long int N1 = N;  // Number of elements in arrays
const long int N2 = N;  // Number of elements in arrays
const long int N3 = N;  // Number of elements in arrays

const int DIM_N = N + 2;





// Kernel that executes on the CUDA device
 __global__ void par( float (*wdtdr)[DIM_N], float *wxm1, float (*dxm1)[DIM_N], int N1, int N2, int DIM_N, int paczka)
 {



  	int idx = blockIdx.x;
    int t1, t2, t3;
    int lb = idx*paczka+1 ;
    int ub =  ((idx+1)*paczka < N1) ? (idx+1)*paczka : N1;


    if (N2 >= 1 && N3 >= 2) {
      for(t1 = lb; t1 <= ub; t1++) {
        for(t2 = 1; t2 <= N2; t2++) {
          //s1(t1,t2,1,22);
          wdtdr[t1][t2] = wdtdr[t1][t2] + wxm1[1]*dxm1[1][t1]*dxm1[1][t2];
          if (t2 <= N2 && N1 >= t1 && t1 >= 1 && t2 >= 1) {
            for(t3 = 2; t3 <= N3; t3++) {
             // s1(t1,t2,t3,22);
               wdtdr[t1][t2] = wdtdr[t1][t2] + wxm1[t3]*dxm1[t3][t1]*dxm1[t3][t2];
            }
          }

        }
      }
    }




}

 __global__ void seq( float (*wdtdr)[DIM_N], float *wxm1,  float (*dxm1)[DIM_N], int N1, int N2, int N3, int DIM_N)
{
	int i, j, ip;
	for(i=1; i<=N1; i++)
	 for(j=1; j<=N2; j++)
	  for(ip=1; ip<=N3; ip++)
	   wdtdr[i][j] = wdtdr[i][j] + wxm1[ip]*dxm1[ip][i]*dxm1[ip][j];


}




  	// main routine that executes on the host
int main(int argc, char **argv)
{
   	  float *wdtdr_h, *wdtdr_hs, *wdtdr_d;  // Pointer to host & device arrays

	  float *wxm1_h, *wxm1_d;  // Pointer to host & device arrays
	  float *dxm1_h, *dxm1_d;  // Pointer to host & device arrays



	  struct timeval s1, f1;
	  double  duration;


  	  size_t size =  DIM_N * DIM_N * sizeof(float);
	  int _size = DIM_N * DIM_N;

  	  wdtdr_h = (float *)malloc(size);        // Allocate array on host
	  wdtdr_hs = (float *)malloc(size);        // Allocate array on host
  	  wxm1_h = (float *)malloc(size);        // Allocate array on host
	  dxm1_h = (float *)malloc(size);        // Allocate array on host

	  // Initialize host array
  	  for (int i=0; i<_size; i++) wdtdr_h[i] = (float)i;
	  for (int i=0; i<_size; i++) wdtdr_hs[i] = (float)i;
  	  for (int i=0; i<_size; i++) wxm1_h[i] = (float)i;
	  for (int i=0; i<_size; i++) dxm1_h[i] = (float)i;

	  //  copy it to CUDA device
	  gettimeofday(&s1, NULL);

	  hipMalloc((void **) &wdtdr_d, size);   // Allocate array on device
	  hipMalloc((void **) &wxm1_d, size);   // Allocate array on device
   	  hipMalloc((void **) &dxm1_d, size);   // Allocate array on device

          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas alloc: %2.6f seconds\n", duration);

	  gettimeofday(&s1, NULL);

	  hipMemcpy(wdtdr_d, wdtdr_hs, size, hipMemcpyHostToDevice);
	  hipMemcpy(wxm1_d, wxm1_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(dxm1_d, dxm1_h, size, hipMemcpyHostToDevice);

          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas copy: %2.6f seconds\n", duration);


	  // SEQ --------------------
          gettimeofday(&s1, NULL);
//          seq<<<1, 1>>>((float(*)[DIM_N])wdtdr_d, wxm1_d, (float(*)[DIM_N])dxm1_d, N1, N2, N3, DIM_N);
          hipDeviceSynchronize();
          gettimeofday(&f1, NULL);
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas sekw.: %2.6f seconds\n", duration);

  	  hipMemcpy(wdtdr_hs, wdtdr_d, sizeof(float)*_size, hipMemcpyDeviceToHost);

	  //------------------------

	  // PAR --------------------

      int block_size = 1;
  	  int n_blocks = atoi(argv[1]);  // CPU
	  int paczka = (int)ceil(N / n_blocks);
      // printf("\n*** %d \n", paczka);

	  hipMemcpy(wdtdr_d, wdtdr_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(wxm1_d, wxm1_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(dxm1_d, dxm1_h, size, hipMemcpyHostToDevice);

      gettimeofday(&s1, NULL);


      par<<< n_blocks, block_size >>> ((float(*)[DIM_N])wdtdr_d, wxm1_d, (float(*)[DIM_N])dxm1_d, N1, N2, DIM_N, paczka);
      hipDeviceSynchronize();
	  gettimeofday(&f1, NULL);


	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas par: %2.6f seconds\n", duration);

	  //------------------------

  	  // Retrieve result from device and store it in host array
      gettimeofday(&s1, NULL);

  	  hipMemcpy(wdtdr_h, wdtdr_d, sizeof(float)*_size, hipMemcpyDeviceToHost);

      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas fetch: %2.6f seconds\n", duration);


	  for (int i=0; i<_size ; i++)
	  if (wdtdr_hs[i] != wdtdr_h[i])
          {
		printf("y %i %2.6f %2.6f\n", i, wdtdr_hs[i], wdtdr_h[i]);
	        printf("blad\n");
		exit(0);
	  }



  	  // Print results
  	  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  	  // Cleanup
  	  free(wdtdr_h); hipFree(wdtdr_d);
	  free(wdtdr_hs);

	  free(wxm1_h); hipFree(wxm1_d);
	  free(dxm1_h); hipFree(dxm1_d);
}
