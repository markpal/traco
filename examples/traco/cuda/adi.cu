#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#include "common.h"

// Kernel that executes on the CUDA device
 __global__ void par1(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i1,j2;

		for (i1=lb; i1<ub; i1++) {
			for (i2=1; i2<N; i2++) {
				X[i1][i2] = X[i1][i2] - X[i1][i2-1] * A[i1][i2] / B[i1][i2-1];
				B[i1][i2] = B[i1][i2] - A[i1][i2] * A[i1][i2] / B[i1][i2-1];
			}
		}
}

 __global__ void par2(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i1,j2;

		for (i1=lb; i1<ub; i1++) {
			X[i1][N-1] = X[i1][N-1] / B[i1][N-1];
		}
}



 __global__ void par3(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i1,j2;

        for (i1=lb; i1<ub; i1++) {
			for (i2=0; i2<N-2; i2++) {
				X[i1][N-i2-2] = (X[i1][N-2-i2] - X[i1][N-2-i2-1] * A[i1][N-i2-3]) / B[i1][N-3-i2];
			}
		}
}

 __global__ void par4(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka+1;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i1,j2;

		for (i1=lb; i1<ub; i1++) {
			for (i2=0; i2<N; i2++) {
				X[i1][i2] = X[i1][i2] - X[i1-1][i2] * A[i1][i2] / B[i1-1][i2];
				B[i1][i2] = B[i1][i2] - A[i1][i2] * A[i1][i2] / B[i1-1][i2];
			}
		}
}

 __global__ void par5(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < N-1) ? (idx+1)*paczka : N-1;
      int i1,j2;

        for (i2=lb; i2<ub; i2++) {
			X[N-1][i2] = X[N-1][i2] / B[N-1][i2];
		}
}

 __global__ void par6(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int paczka)
{
  	  int idx = blockIdx.x;
      int lb = idx*paczka;
      int ub =  ((idx+1)*paczka < N-2) ? (idx+1)*paczka : N-2;
      int i1,j2;

		for (i1=lb; i1<ub; i1++) {
			for (i2=0; i2<N; i2++) {
				X[N-2-i1][i2] = (X[N-2-i1][i2] - X[N-i1-3][i2] * A[N-3-i1][i2]) / B[N-2-i1][i2];
			}
		}
}




 __global__ void seq(float (*A)[N][N], float (*B)[N][N], float (*X)[N][N], int N, int TSTEPS, int paczka)
{
    int i,j,t,i1,i2;

    for (t=0; t<TSTEPS; t++) {
		for (i1=0; i1<N; i1++) {
			for (i2=1; i2<N; i2++) {
				X[i1][i2] = X[i1][i2] - X[i1][i2-1] * A[i1][i2] / B[i1][i2-1];
				B[i1][i2] = B[i1][i2] - A[i1][i2] * A[i1][i2] / B[i1][i2-1];
			}
		}
		for (i1=0; i1<N; i1++) {
			X[i1][N-1] = X[i1][N-1] / B[i1][N-1];
		}
		for (i1=0; i1<N; i1++) {
			for (i2=0; i2<N-2; i2++) {
				X[i1][N-i2-2] = (X[i1][N-2-i2] - X[i1][N-2-i2-1] * A[i1][N-i2-3]) / B[i1][N-3-i2];
			}
		}
		for (i1=1; i1<N; i1++) {
			for (i2=0; i2<N; i2++) {
				X[i1][i2] = X[i1][i2] - X[i1-1][i2] * A[i1][i2] / B[i1-1][i2];
				B[i1][i2] = B[i1][i2] - A[i1][i2] * A[i1][i2] / B[i1-1][i2];
			}
		}
		for (i2=0; i2<N; i2++) {
			X[N-1][i2] = X[N-1][i2] / B[N-1][i2];
		}
		for (i1=0; i1<N-2; i1++) {
			for (i2=0; i2<N; i2++) {
				X[N-2-i1][i2] = (X[N-2-i1][i2] - X[N-i1-3][i2] * A[N-3-i1][i2]) / B[N-2-i1][i2];
			}
		}
	}
}

  	// main routine that executes on the host
int main(int argc, char **argv)
{
  	  int t,i,j;

  	  float *A_h, *A_d;  // Pointer to host & device arrays
  	  float *X_h, *X_d;  // Pointer to host & device arrays
  	  float *B_h, *B_d;  // Pointer to host & device arrays
  	  float *B_hs, *B_ds;  // Pointer to host & device arrays
  	  float *X_hs, *B_ds;  // Pointer to host & device arrays

	  struct timeval s1, f1;
	  double  duration;

  	  size_t size =  N * N * sizeof(float);
	  int _size = N * N * LENGTH;


  	  A_h = (float *)malloc(size);        // Allocate array on host
  	  B_h = (float *)malloc(size);        // Allocate array on host
  	  B_hs = (float *)malloc(size);        // Allocate array on host
      X_h = (float *)malloc(size);        // Allocate array on host
  	  X_hs = (float *)malloc(size);        // Allocate array on host

        // Set the input data
	for (i=0; i<N; i++) {
		for (j=0; j<N; j++) {
			X[i][j] = ((float) i*(j+1) + 1) / N;
			A[i][j] = ((float) i*(j+2) + 2) / N;
			B[i][j] = ((float) i*(j+3) + 3) / N;
		}
	}

	  hipMalloc((void **) &X_ds, size);   // Allocate array on device
	  hipMalloc((void **) &B_ds, size);   // Allocate array on device

	  hipMemcpy(X_ds, X_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(B_ds, B_h, size, hipMemcpyHostToDevice);

	  //  copy it to CUDA device
	  gettimeofday(&s1, NULL);

	  hipMalloc((void **) &X_d, size);   // Allocate array on device
	  hipMalloc((void **) &A_d, size);   // Allocate array on device
	  hipMalloc((void **) &B_d, size);   // Allocate array on device

      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas alloc: %2.6f seconds\n", duration);

	  gettimeofday(&s1, NULL);
  	  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  	  hipMemcpy(X_d, X_h, size, hipMemcpyHostToDevice);
  	  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas copy: %2.6f seconds\n", duration);


	  // SEQ --------------------
      gettimeofday(&s1, NULL);
      seq<<<1, 1>>>((float(*)[N][N])A_ds, (float(*)[N][N]B_ds, (float(*)[N][N]X_ds, N, TSTEPS, paczka)
      hipDeviceSynchronize();
      gettimeofday(&f1, NULL);

	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas sekw.: %2.6f seconds\n", duration);

  	  hipMemcpy(X_hs, X_ds, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(B_hs, B_ds, sizeof(float)*size, hipMemcpyDeviceToHost);

	  hipFree(X_ds);
	  hipFree(B_ds);


	  //------------------------

	  // PAR --------------------

      int block_size = 1;
  	  int n_blocks = atoi(argv[1]);  // CPU
	  int paczka = (int)ceil(N / n_blocks);
         // printf("\n*** %d \n", paczka);

      gettimeofday(&s1, NULL);
	  for (t=0; t<ITER; t++) {
        par1<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, (float(*)[N][N]X_d, N, paczka);
        hipDeviceSynchronize();
        par2<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, (float(*)[N][N]X_d, N, paczka);
        hipDeviceSynchronize();
        par3<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, (float(*)[N][N]X_d, N, paczka);
        hipDeviceSynchronize();
        par4<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, (float(*)[N][N]X_d, N, paczka);
        hipDeviceSynchronize();
        par5<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, (float(*)[N][N]X_d, N, paczka);
        hipDeviceSynchronize();
        par6<<< n_blocks, block_size >>>((float(*)[N][N])A_d, (float(*)[N][N]B_d, (float(*)[N][N]X_d, N, paczka);
        hipDeviceSynchronize();
	  }
	  hipDeviceSynchronize();
      gettimeofday(&f1, NULL);


	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas par: %2.6f seconds\n", duration);

	  //------------------------

  	  // Retrieve result from device and store it in host array
      gettimeofday(&s1, NULL);
      //hipMemcpy(x_h, x_d, sizeof(float)*size, hipMemcpyDeviceToHost);
      hipMemcpy(X_h, X_d, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(B_h, B_d, sizeof(float)*size, hipMemcpyDeviceToHost);
	  //hipMemcpy(twiddle_h, twiddle_d, sizeof(float)*N, hipMemcpyDeviceToHost);
      gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas fetch: %2.6f seconds\n", duration);

/*
	  for (int i=0; i<_size ; i++)
	  if (y_hs[i] != y_h[i])
          {
		printf("%i %2.6f %2.6f\n", i, y_hs[i], y_h[i]);
	        printf("blad\n");
		exit(0);
	  }
*/

  	  // Print results
  	  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  	  // Cleanup
      hipFree(A_d);
      hipFree(B_d);
      hipFree(X_d);

      free(A_h);
      free(B_h);
      free(X_h);

}
