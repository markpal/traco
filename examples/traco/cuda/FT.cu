#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

const long int N = 200;  // Number of elements in arrays
const long int N1 = N;  // Number of elements in arrays
const long int N2 = N;  // Number of elements in arrays
const long int N3 = N;  // Number of elements in arrays

const int DIM_N = N + 2;

// Kernel that executes on the CUDA device
 __global__ void par(float (*x)[DIM_N][DIM_N], float (*y)[DIM_N][DIM_N], float (*twiddle)[DIM_N][DIM_N], int N1, int N2, int N3, int DIM_N, int paczka)
{
  	  int idx = blockIdx.x;
          int t1, t2, t3;
          int lb = idx*paczka+1;
          int ub =  ((idx+1)*paczka < N1) ? (idx+1)*paczka : N1;

	if (N3 >= 1 && N2 >= 1) {
	  for(t1 = lb; t1 <= ub; t1++) {
	
	      for(t3 = 1; t3 <= N3; t3++) {
	          for(t2 = 1; t2 <= N2; t2++) {
		y[t2][t3][t1] = y[t2][t3][t1] + twiddle[t2][t3][t1];
		if (t3 >= 1 && N3 >= t3 && t2 >= 1 && N2 >= t2 && t1 >= 1 && N1 >= t1) {
		  y[t2][t3][t1] = x[t2][t3][t1];
		}

	      }
	    }
	  }
	}


}

 __global__ void seq(float (*x)[DIM_N][DIM_N], float (*y)[DIM_N][DIM_N], float (*twiddle)[DIM_N][DIM_N], int N1, int N2, int N3, int DIM_N)
{
	int i,j, k;
	for(i=1; i<=N1; i++)
          for(k=1; k<=N2; k++)
	  for(j=1; j<=N3; j++)

              {
		   y[j][k][i] = y[j][k][i] + twiddle[j][k][i];
		   y[j][k][i] = x[j][k][i];
	      }
}

  	// main routine that executes on the host
int main(int argc, char **argv)
{
  	  float *y_h, *y_d;  // Pointer to host & device arrays
   	  float *y_hs, *y_ds;  // Pointer to host & device arrays
   	  float *x_hs, *x_ds;  // Pointer to host & device arrays

	  float *x_h, *x_d;  // Pointer to host & device arrays
	  float *twiddle_h, *twiddle_d;  // Pointer to host & device arrays

	  struct timeval s1, f1;
	  double  duration;

  	  size_t size =  DIM_N * DIM_N * DIM_N * sizeof(float);
	  int _size = DIM_N * DIM_N * DIM_N;

  	  y_h = (float *)malloc(size);        // Allocate array on host
	  x_h = (float *)malloc(size);        // Allocate array on host
  	  y_hs = (float *)malloc(size);        // Allocate array on host
  	  x_hs = (float *)malloc(size);        // Allocate array on host
	  twiddle_h = (float *)malloc(size);        // Allocate array on host

	  // Initialize host array
  	  for (int i=0; i<_size; i++) x_h[i] = (float)i;

	  hipMalloc((void **) &y_ds, size);   // Allocate array on device
	  hipMalloc((void **) &x_ds, size);   // Allocate array on device

	  hipMemcpy(y_ds, y_h, size, hipMemcpyHostToDevice);
	  	  hipMemcpy(x_ds, x_h, size, hipMemcpyHostToDevice);

	  //  copy it to CUDA device
	  gettimeofday(&s1, NULL);

	  hipMalloc((void **) &x_d, size);   // Allocate array on device
	  hipMalloc((void **) &y_d, size);   // Allocate array on device
	  hipMalloc((void **) &twiddle_d, size);   // Allocate array on device

          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas alloc: %2.6f seconds\n", duration);

	  gettimeofday(&s1, NULL);
  	  hipMemcpy(x_d, x_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(y_d, y_h, size, hipMemcpyHostToDevice);
          hipMemcpy(twiddle_d, twiddle_h, size, hipMemcpyHostToDevice);

          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas copy: %2.6f seconds\n", duration);


	  // SEQ --------------------
          gettimeofday(&s1, NULL);
          seq<<<1, 1>>>((float(*)[DIM_N][DIM_N])x_ds, (float(*)[DIM_N][DIM_N])y_ds, (float(*)[DIM_N][DIM_N])twiddle_d, N1, N2, N3, DIM_N);
          hipDeviceSynchronize();
          gettimeofday(&f1, NULL);
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas sekw.: %2.6f seconds\n", duration);

  	  hipMemcpy(y_hs, y_ds, sizeof(float)*size, hipMemcpyDeviceToHost);
  	  hipMemcpy(x_hs, x_ds, sizeof(float)*size, hipMemcpyDeviceToHost);
	  hipFree(x_ds);
	  
	  //------------------------

	  // PAR --------------------

          int block_size = 1;
  	  int n_blocks = atoi(argv[1]);  // CPU
	  int paczka = (int)ceil(N / n_blocks);
         // printf("\n*** %d \n", paczka);

          gettimeofday(&s1, NULL);
	  par<<< n_blocks, block_size >>> ((float(*)[DIM_N][DIM_N])x_d, (float(*)[DIM_N][DIM_N])y_d, (float(*)[DIM_N][DIM_N])twiddle_d, N1, N2, N3, DIM_N, paczka);
          hipDeviceSynchronize();
          gettimeofday(&f1, NULL);


	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas par: %2.6f seconds\n", duration);

	  //------------------------

  	  // Retrieve result from device and store it in host array
          gettimeofday(&s1, NULL);
          //hipMemcpy(x_h, x_d, sizeof(float)*size, hipMemcpyDeviceToHost);
	  hipMemcpy(y_h, y_d, sizeof(float)*size, hipMemcpyDeviceToHost);
	  //hipMemcpy(twiddle_h, twiddle_d, sizeof(float)*N, hipMemcpyDeviceToHost);
          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas fetch: %2.6f seconds\n", duration);


	  for (int i=0; i<_size ; i++)
	  if (y_hs[i] != y_h[i])
          {
		printf("%i %2.6f %2.6f\n", i, y_hs[i], y_h[i]);
	        printf("blad\n");
		exit(0);
	  }


  	  // Print results
  	  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  	  // Cleanup
  	  free(x_h); hipFree(x_d);
	  free(y_h); hipFree(y_d);
          free(twiddle_h); hipFree(twiddle_d);
	  free(y_hs);
}
