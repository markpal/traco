#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

const long int N = 2048;  // Number of elements in arrays
const long int N1 = N;  // Number of elements in arrays
const long int N2 = N;  // Number of elements in arrays
const long int N3 = 10;  // Number of elements in arrays
const long int N4 = 10;  // Number of elements in arrays

const int DIM_N = N + 2;

// Kernel that executes on the CUDA device
 __global__ void par(float (*tm2)[DIM_N][DIM_N], float (*u)[DIM_N][DIM_N], float (*wdtdr)[DIM_N], int N1, int N2, int N3, int N4, int DIM_N, int paczka)
{
  	  int idx = blockIdx.x;
          int t1, t3, t4, t2;
          int lb = idx*paczka+1;
          int ub =  ((idx+1)*paczka < N1) ? (idx+1)*paczka : N1;


if (N3 >= 1 && N2 >= 2 && N4 >= 1) {
  for(t1 = lb; t1 <= ub; t1++) {
    for(t3 = 1; t3 <= N3; t3++) {
      for(t4 = 1; t4 <= N4; t4++) {
        //s1(t1,1,t3,t4,24);
        tm2[t4][t3][t1] =  tm2[t4][t3][t1] + u[t4][1][t3] * wdtdr[1][t3];
        if (t1 >= 1 && t4 >= 1 && t4 <= N4 && t3 >= 1 && t3 <= N3 && N1 >= t1) {
          for(t2 = 2; t2 <= N2; t2++) {
           // s1(t1,t2,t3,t4,24);
            tm2[t4][t3][t1] =  tm2[t4][t3][t1] + u[t4][t2][t3] * wdtdr[t2][t3];
          }
        }

      }
    }
  }
}


}




 __global__ void seq(float (*tm2)[DIM_N][DIM_N], float (*u)[DIM_N][DIM_N], float (*wdtdr)[DIM_N], int N1, int N2, int N3, int N4, int DIM_N)
{
	int i,j, k, iz;
	for(iz=1; iz<=N1; iz++)
	  for(k=1; k<=N2; k++)
        for(j=1; j<=N3; j++)
		   for(i=1; i<=N4; i++)
           {
			//   tm2[i*DIM_N*DIM_N + j*DIM_N + iz] = 7;
			   //tm2[0] = (float)8;

			   tm2[i][j][iz] =  tm2[i][j][iz] + u[i][k][j] * wdtdr[k][j];
	       }
	  //     tm2[0] = (float)8;
	     //  tm2[1] = (float)8;
}

  	// main routine that executes on the host
int main(int argc, char **argv)
{
  	  float *tm2_h, *tm2_d;  // Pointer to host & device arrays
   	  float *tm2_hs, *tm2_ds;  // Pointer to host & device arrays

	  float *u_h, *u_d;  // Pointer to host & device arrays
	  float *wdtdr_h, *wdtdr_d;  // Pointer to host & device arrays



	  struct timeval s1, f1;
	  double  duration;

  	  size_t size =  DIM_N * DIM_N * N3 *N4 * sizeof(float);
	  int _size = DIM_N * DIM_N * N3 *N4;


  	  tm2_h = (float *)malloc(size);        // Allocate array on host
	  u_h = (float *)malloc(size);        // Allocate array on host
  	  tm2_hs = (float *)malloc(size);        // Allocate array on host
	  wdtdr_h = (float *)malloc(size);        // Allocate array on host

	  // Initialize host array
  	  for (int i=0; i<_size; i++) tm2_h[i] = (float)i;
	  for (int i=0; i<_size; i++) tm2_hs[i] = (float)i;
	  for (int i=0; i<_size; i++) u_h[i] = (float)i;
	  for (int i=0; i<_size; i++) wdtdr_h[i] = (float)i;

	  hipMalloc((void **) &tm2_ds, size);   // Allocate array on device

	  hipMemcpy(tm2_ds, tm2_hs, size, hipMemcpyHostToDevice);

	  //  copy it to CUDA device
	  gettimeofday(&s1, NULL);

	 hipMalloc((void **) &tm2_d, size);   // Allocate array on device
	 hipMalloc((void **) &u_d, size);   // Allocate array on device
	 hipMalloc((void **) &wdtdr_d, size);   // Allocate array on device

         gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas alloc: %2.6f seconds\n", duration);

    gettimeofday(&s1, NULL);
  	  hipMemcpy(tm2_d, tm2_h, size, hipMemcpyHostToDevice);
	  hipMemcpy(u_d, u_h, size, hipMemcpyHostToDevice);
          hipMemcpy(wdtdr_d, wdtdr_h, size, hipMemcpyHostToDevice);

          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas send: %2.6f seconds\n", duration);


	  // SEQ --------------------
          gettimeofday(&s1, NULL);
//          seq<<<1, 1>>>((float(*)[DIM_N][DIM_N])tm2_ds, (float(*)[DIM_N][DIM_N])u_d, (float(*)[DIM_N])wdtdr_d, N1, N2, N3, N4, DIM_N);
          hipDeviceSynchronize();
          gettimeofday(&f1, NULL);

	  hipMemcpy(tm2_hs, tm2_ds, sizeof(float)*_size, hipMemcpyDeviceToHost);
	 // printf("%i %2.6f %2.6f\n", 0, tm2_hs[0], tm2_h[0]);
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas sekw.: %2.6f seconds\n", duration);

	  //exit(0);
	  //------------------------

	  // PAR --------------------

          int block_size = 1;
  	  int n_blocks = atoi(argv[1]);  // CPU
	  int paczka = (int)ceil(N / n_blocks);
         // printf("\n*** %d \n", paczka);

          gettimeofday(&s1, NULL);
		par<<< n_blocks, block_size >>> ((float(*)[DIM_N][DIM_N])tm2_d, (float(*)[DIM_N][DIM_N])u_d, (float(*)[DIM_N])wdtdr_d, N1, N2, N3, N4, DIM_N, paczka);
          hipDeviceSynchronize();
          gettimeofday(&f1, NULL);


	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas par: %2.6f seconds\n", duration);

	  //------------------------

  	  // Retrieve result from device and store it in host array
          gettimeofday(&s1, NULL);
          //hipMemcpy(x_h, x_d, sizeof(float)*size, hipMemcpyDeviceToHost);
	  hipMemcpy(tm2_h, tm2_d, sizeof(float)*_size, hipMemcpyDeviceToHost);
	  //hipMemcpy(wdtdr_h, wdtdr_d, sizeof(float)*N, hipMemcpyDeviceToHost);
          gettimeofday(&f1, NULL);
  	  // Do calculation on device:
	  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	  printf("Czas fetch: %2.6f seconds\n", duration);


	  for (int i=0; i<_size ; i++)
	  if (tm2_hs[i] != tm2_h[i])
          {
		printf("%i %2.6f %2.6f\n", i, tm2_hs[i], tm2_h[i]);
	        printf("blad\n");
		exit(0);
	  }


  	  // Print results
  	  //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  	  // Cleanup
  	  free(tm2_h); hipFree(tm2_d);
	  free(u_h); hipFree(u_d);
          free(wdtdr_h); hipFree(wdtdr_d);
	  free(tm2_hs); hipFree(tm2_ds);
}
