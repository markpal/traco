#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#define N 256

const long int N1 = N;  // Number of elements in arrays
const long int N2 = N;  // Number of elements in arrays
const long int N3 = N;  // Number of elements in arrays

const int DIM_N = N + 2;


// Kernel that executes on the CUDA device
 __global__ void par(float (*rho_i)[DIM_N][DIM_N], float (*u)[6][DIM_N][DIM_N], float (*rsd)[6][DIM_N][DIM_N], float (*frct)[6][DIM_N][DIM_N], float (*qs)[DIM_N][DIM_N], int paczka)
{
      int idx = blockIdx.x;
      int k, m, i, j;
      float tmp;
      int lb = idx*paczka+1;
      int ub =  ((idx+1)*paczka < N1) ? (idx+1)*paczka : N1;


   for(k = lb; k <= ub; k += 1)
      for(j = 1; j <= N2; j += 1)
         for(i = 1; i <= N3; i += 1) {
            for(m = 1; m <= 5; m += 1)
               rsd[m][i][j][k] = -frct[m][i][j][k];
            tmp = 1.0/u[1][i][j][k];
            rho_i[i][j][k] = tmp;
            qs[i][j][k] = 0.5*(u[2][i][j][k]*u[2][i][j][k]+u[3][i][j][k]*u[3][i][j][k]+u[4][i][j][k]*u[4][i][j][k])*tmp;
         }

}



__global__ void seq(float (*rho_i)[DIM_N][DIM_N], float (*u)[6][DIM_N][DIM_N], float (*rsd)[6][DIM_N][DIM_N], float (*frct)[6][DIM_N][DIM_N], float (*qs)[DIM_N][DIM_N])
{
	int k, i, j, m;
	float tmp;


   for(k = 1; k <= N1; k += 1)
      for(j = 1; j <= N2; j += 1)
         for(i = 1; i <= N3; i += 1) {
            for(m = 1; m <= 5; m += 1)
               rsd[m][i][j][k] = -frct[m][i][j][k];
            tmp = 1.0/u[1][i][j][k];
            rho_i[i][j][k] = tmp;
            qs[i][j][k] = 0.5*(u[2][i][j][k]*u[2][i][j][k]+u[3][i][j][k]*u[3][i][j][k]+u[4][i][j][k]*u[4][i][j][k])*tmp;
         }

}



int main(int argc, char **argv)
{
    float *rho_i_h, *rho_i_d;  // Pointer to host & device arrays
    float *rho_i_hs, *rho_i_ds;  // Pointer to host & device arrays

    float *u_h, *u_d;  // Pointer to host & device arrays

    float *rsd_h, *rsd_d;  // Pointer to host & device arrays
    float *rsd_hs, *rsd_ds;  // Pointer to host & device arrays

    float *frct_h, *frct_d;  // Pointer to host & device arrays

    float *qs_h, *qs_d;  // Pointer to host & device arrays
    float *qs_hs, *qs_ds;  // Pointer to host & device arrays



    struct timeval s1, f1;
	double  duration;

  	size_t size =  DIM_N * DIM_N * DIM_N * sizeof(float);
	int _size = DIM_N * DIM_N * DIM_N;

  	rho_i_h = (float *)malloc(size);        // Allocate array on host
  	rho_i_hs = (float *)malloc(size);        // Allocate array on host
  	qs_h = (float *)malloc(size);
  	qs_hs = (float *)malloc(size);

  	u_h = (float *)malloc(size*6);
  	rsd_h = (float *)malloc(size*6);
  	rsd_hs = (float *)malloc(size*6);
  	frct_h = (float *)malloc(size*6);

	// Initialize host array
  	for (int i=0; i<_size*6; i++) u_h[i] = (float)i;
  	for (int i=0; i<_size*6; i++) frct_h[i] = (float)i;
  	for (int i=0; i<_size*6; i++) rsd_h[i] = (float)i;
  	for (int i=0; i<_size*6; i++) rsd_hs[i] = (float)i;
  	for (int i=0; i<_size; i++) qs_h[i] = (float)i;
  	for (int i=0; i<_size; i++) qs_hs[i] = (float)i;

	//  hipMalloc((void **) &pdiff_ds, size);   // Allocate array on device
	hipMalloc((void **) &rho_i_ds, size);   // Allocate array on device
	hipMalloc((void **) &rsd_ds, size*6);   // Allocate array on device
    hipMalloc((void **) &qs_ds, size);   // Allocate array on device


    //hipMemcpy(rms_ds, rms_hs, sizeof(float)*6, hipMemcpyHostToDevice);
    //hipMemcpy(qs_ds, qs_hs, size, hipMemcpyHostToDevice);


	//  copy it to CUDA device
	gettimeofday(&s1, NULL);
	hipMalloc((void **) &rho_i_d, size);   // Allocate array on device
    hipMalloc((void **) &qs_d, size);   // Allocate array on device
    hipMalloc((void **) &u_d, 6*size);   // Allocate array on device
    hipMalloc((void **) &frct_d, 6*size);   // Allocate array on device
    hipMalloc((void **) &rsd_d, 6*size);   // Allocate array on device
    gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas alloc: %2.6f seconds", duration);


    gettimeofday(&s1, NULL);
  	hipMemcpy(u_d, u_h, size*6, hipMemcpyHostToDevice);
  	hipMemcpy(frct_d, frct_h, size*6, hipMemcpyHostToDevice);
    gettimeofday(&f1, NULL);

  	 // Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("\nCzas send: %2.6f seconds\n", duration);

      // SEQ --------------------
    gettimeofday(&s1, NULL);
    seq<<<1, 1>>>((float(*)[DIM_N][DIM_N])rho_i_ds, (float(*)[6][DIM_N][DIM_N])u_d, (float(*)[6][DIM_N][DIM_N])rsd_ds, (float(*)[6][DIM_N][DIM_N])frct_d, (float(*)[DIM_N][DIM_N])qs_ds);
    hipDeviceSynchronize();
    gettimeofday(&f1, NULL);

	// hipMemcpy(pdiff_hs, pdiff_ds, sizeof(int)*_size, hipMemcpyDeviceToHost);
	hipMemcpy(rho_i_hs, rho_i_ds, size, hipMemcpyDeviceToHost);
	hipMemcpy(rsd_hs, rsd_ds, size*6, hipMemcpyDeviceToHost);
	hipMemcpy(qs_hs, qs_ds, size, hipMemcpyDeviceToHost);


	// printf("%i %2.6f %2.6f\n", 0, tm2_hs[0], tm2_h[0]);
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas sekw.: %2.6f seconds\n", duration);

	//------------------------

	// PAR --------------------

    int block_size = 1;
  	int n_blocks = atoi(argv[1]);  // CPU
	int paczka = (int)ceil(N1 / n_blocks);
printf("%i", paczka);

    gettimeofday(&s1, NULL);
    par<<< n_blocks, block_size >>> ((float(*)[DIM_N][DIM_N])rho_i_d, (float(*)[6][DIM_N][DIM_N])u_d, (float(*)[6][DIM_N][DIM_N])rsd_d, (float(*)[6][DIM_N][DIM_N])frct_d, (float(*)[DIM_N][DIM_N])qs_d, paczka);
    hipDeviceSynchronize();
    gettimeofday(&f1, NULL);


	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas par: %2.6f seconds\n", duration);

    //------------------------

	//------------------------

  	// Retrieve result from device and store it in host array
    gettimeofday(&s1, NULL);
	hipMemcpy(rho_i_h, rho_i_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(rsd_h, rsd_d, size*6, hipMemcpyDeviceToHost);
	hipMemcpy(qs_h, qs_d, size, hipMemcpyDeviceToHost);
	gettimeofday(&f1, NULL);
  	// Do calculation on device:
	duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
	printf("Czas fetch: %2.6f seconds\n", duration);


	for (int i=0; i<_size ; i++)
          if (qs_hs[i] != qs_h[i])
          {
            printf("%i %f %f\n", i, qs_hs[i], qs_h[i]);
            printf("blad\n");
            exit(0);
          }

    return 0;
}





