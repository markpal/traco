#include "hip/hip_runtime.h"
#include <assert.h>
#include "atax_kernel.hu"
#include<sys/time.h>
#include<stdlib.h>
#include<stdio.h>
#include <math.h>

#define ceild(n,d)  ceil(((double)(n))/((double)(d)))
#define floord(n,d) floor(((double)(n))/((double)(d)))
#define max(x,y)    ((x) > (y)? (x) : (y))
#define min(x,y)    ((x) < (y)? (x) : (y))


int N = 1024;
int DIM;
int NY, NX;
int DIM1;
int DIM2;
int DIM3;





void seq(float A[DIM1][DIM2], float y[DIM1], float tmp[DIM1], float x[DIM1]) {
int i,j,k, alpha=0;

if (NY >= 1 || NX >= 1)
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float *dev_y;
    float *dev_A;
    float *dev_tmp;
    float *dev_x;
    
    cudaCheckReturn(hipMalloc((void **) &dev_y, (NY) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_A, (NX) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_tmp, (NX) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_x, (NY) * sizeof(float)));
    
cudaCheckReturn(hipMemcpy(dev_A, A, (NX) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_x, x, (NY) * sizeof(float), hipMemcpyHostToDevice));
    
    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    {
      if (NX >= 1) {
        {
          dim3 k0_dimBlock(32);
          dim3 k0_dimGrid((1048512 - NX >= 0) ? ((NX + 31) / 32) : (-1048513 + NX >= 0 && 1048544 - NX >= 0) ? (32767) : 32768);
          kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_tmp, DIM2, NX, NY);
          cudaCheckKernel();
        }
        
        if (NY >= 1)
          {
            dim3 k1_dimBlock(32);
            dim3 k1_dimGrid((1048512 - NX >= 0) ? ((NX + 31) / 32) : (-1048513 + NX >= 0 && 1048544 - NX >= 0) ? (32767) : 32768);
            kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_tmp, dev_x, DIM2, NX, NY);
            cudaCheckKernel();
          }
          
      }
      if (NY >= 1) {
        {
          dim3 k2_dimBlock(32);
          dim3 k2_dimGrid((1048512 - NY >= 0) ? ((NY + 31) / 32) : (-1048513 + NY >= 0 && 1048544 - NY >= 0) ? (32767) : 32768);
          kernel2 <<<k2_dimGrid, k2_dimBlock>>> (dev_y, DIM2, NX, NY);
          cudaCheckKernel();
        }
        
        if (NX >= 1)
          {
            dim3 k3_dimBlock(32);
            dim3 k3_dimGrid((1048512 - NY >= 0) ? ((NY + 31) / 32) : (-1048513 + NY >= 0 && 1048544 - NY >= 0) ? (32767) : 32768);
            kernel3 <<<k3_dimGrid, k3_dimBlock>>> (dev_y, dev_A, dev_tmp, DIM2, NX, NY);
            cudaCheckKernel();
          }
          
      }
    }
cudaCheckReturn(hipMemcpy(y, dev_y, (NY) * sizeof(float), hipMemcpyDeviceToHost));
cudaCheckReturn(hipMemcpy(tmp, dev_tmp, (NX) * sizeof(float), hipMemcpyDeviceToHost));
    
cudaCheckReturn(hipFree(dev_y));
cudaCheckReturn(hipFree(dev_A));
cudaCheckReturn(hipFree(dev_tmp));
cudaCheckReturn(hipFree(dev_x));
  }

}





int main(int argc, char *argv[]) {

  // number of processors
  int tool_kind=1, num_proc=1;

  tool_kind = atoi(argv[1]);
  num_proc = atoi(argv[2]);

  if(argc > 3)
    N = atoi(argv[3]);

  NY = NX = N;
  DIM = DIM1 = DIM2 = DIM3 = N+20;

  float (*A)[DIM2], *x, *y, *tmp;

  A = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  x = (float (*))malloc(DIM1 * sizeof(float));
  y = (float (*))malloc(DIM1 * sizeof(float));
  tmp = (float (*))malloc(DIM1 * sizeof(float));


  // variables to meassure time
  struct timeval s1, f1;
  double  duration, duration1;

  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  gettimeofday(&s1, NULL);

    seq(A,y,tmp,x);

  gettimeofday(&f1, NULL);

  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
  printf("Czas: %2.3f seconds\n", duration);
  // -----------------------------------
  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!





}









