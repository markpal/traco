#include "hip/hip_runtime.h"
#include "atax_kernel.hu"
__global__ void kernel0(float *tmp, int DIM2, int NX, int NY)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int g1 = 32 * b0; g1 < NX; g1 += 1048576) {
      if (NX >= t0 + g1 + 1)
        tmp[t0 + g1] = (0);
      __syncthreads();
    }
}
__global__ void kernel1(float *A, float *tmp, float *x, int DIM2, int NX, int NY)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    float private_tmp[1];
    __shared__ float shared_x[32];

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NX; g1 += 1048576) {
      if (NX >= t0 + g1 + 1)
        private_tmp[0] = tmp[t0 + g1];
      for (int g5 = 0; g5 < NY; g5 += 32) {
        if (NY >= t0 + g5 + 1)
          shared_x[t0] = x[t0 + g5];
        if (DIM2 >= t0 + g5 + 1)
          for (int c0 = 0; c0 <= min(31, NX - g1 - 1); c0 += 1)
            shared_A[c0][t0] = A[(g1 + c0) * (DIM2) + (t0 + g5)];
        __syncthreads();
        if (NX >= t0 + g1 + 1)
          for (int c2 = 0; c2 <= min(31, NY - g5 - 1); c2 += 1)
            private_tmp[0] = (private_tmp[0] + (shared_A[t0][c2] * shared_x[c2]));
        __syncthreads();
      }
      if (NX >= t0 + g1 + 1)
        tmp[t0 + g1] = private_tmp[0];
      __syncthreads();
    }
}
__global__ void kernel2(float *y, int DIM2, int NX, int NY)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int g1 = 32 * b0; g1 < NY; g1 += 1048576) {
      if (NY >= t0 + g1 + 1)
        y[t0 + g1] = (0);
      __syncthreads();
    }
}
__global__ void kernel3(float *y, float *A, float *tmp, int DIM2, int NX, int NY)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    float private_y[1];
    __shared__ float shared_tmp[32];

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NY; g1 += 1048576) {
      if (NY >= t0 + g1 + 1)
        private_y[0] = y[t0 + g1];
      for (int g5 = 0; g5 < NX; g5 += 32) {
        if (NX >= t0 + g5 + 1)
          shared_tmp[t0] = tmp[t0 + g5];
        __syncthreads();
        if (NY >= t0 + g1 + 1)
          for (int c2 = 0; c2 <= min(31, NX - g5 - 1); c2 += 1)
            private_y[0] = (private_y[0] + (A[(g5 + c2) * (DIM2) + t0 + g1] * shared_tmp[c2]));
        __syncthreads();
      }
      if (NY >= t0 + g1 + 1)
        y[t0 + g1] = private_y[0];
      __syncthreads();
    }
}
