#include "hip/hip_runtime.h"
#include <assert.h>
#include "2mm_kernel.hu"
#include<stdlib.h>
#include<stdio.h>
#include <math.h>
#include<sys/time.h>

#define ceild(n,d)  ceil(((double)(n))/((double)(d)))
#define floord(n,d) floor(((double)(n))/((double)(d)))
#define max(x,y)    ((x) > (y)? (x) : (y))
#define min(x,y)    ((x) < (y)? (x) : (y))


int N = 1024;
int DIM;
int NI, NJ, NL, NM, NK;
int DIM1;
int DIM2;
int DIM3;





void seq(float A[DIM1][DIM2], float B[DIM1][DIM2], float C[DIM1][DIM2], float D[DIM1][DIM2], float tmp[DIM1][DIM2]) {
int i,j,k, alpha=0, beta = 0;

if ((NJ >= 1 && NI >= 1) || (NL >= 1 && NI >= 1))
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float *dev_tmp;
    float *dev_A;
    float *dev_C;
    float *dev_B;
    float *dev_D;
    
    cudaCheckReturn(hipMalloc((void **) &dev_tmp, (NI) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_A, (NI) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_C, (NJ) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (NK) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_D, (NI) * (DIM2) * sizeof(float)));
    
cudaCheckReturn(hipMemcpy(dev_tmp, tmp, (NI) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_A, A, (NI) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_C, C, (NJ) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_B, B, (NK) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_D, D, (NI) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
    
    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    {
      if (NJ >= 1) {
        {
          dim3 k0_dimBlock(16, 32);
          dim3 k0_dimGrid((8128 - NJ >= 0) ? ((NJ + 31) / 32) : (-8129 + NJ >= 0 && 8160 - NJ >= 0) ? (255) : 256, (8128 - NI >= 0) ? ((NI + 31) / 32) : (-8129 + NI >= 0 && 8160 - NI >= 0) ? (255) : 256);
          kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_tmp, DIM2, NI, NL, NJ, NK);
          cudaCheckKernel();
        }
        
        if (NK >= 1)
          {
            dim3 k1_dimBlock(16, 32);
            dim3 k1_dimGrid((8128 - NJ >= 0) ? ((NJ + 31) / 32) : (-8129 + NJ >= 0 && 8160 - NJ >= 0) ? (255) : 256, (8128 - NI >= 0) ? ((NI + 31) / 32) : (-8129 + NI >= 0 && 8160 - NI >= 0) ? (255) : 256);
            kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_tmp, alpha, dev_A, dev_B, DIM2, NI, NL, NJ, NK);
            cudaCheckKernel();
          }
          
      }
      if (NL >= 1) {
        {
          dim3 k2_dimBlock(16, 32);
          dim3 k2_dimGrid((8128 - NL >= 0) ? ((NL + 31) / 32) : (-8129 + NL >= 0 && 8160 - NL >= 0) ? (255) : 256, (8128 - NI >= 0) ? ((NI + 31) / 32) : (-8129 + NI >= 0 && 8160 - NI >= 0) ? (255) : 256);
          kernel2 <<<k2_dimGrid, k2_dimBlock>>> (beta, dev_D, DIM2, NI, NL, NJ, NK);
          cudaCheckKernel();
        }
        
        if (NJ >= 1)
          {
            dim3 k3_dimBlock(16, 32);
            dim3 k3_dimGrid((8128 - NL >= 0) ? ((NL + 31) / 32) : (-8129 + NL >= 0 && 8160 - NL >= 0) ? (255) : 256, (8128 - NI >= 0) ? ((NI + 31) / 32) : (-8129 + NI >= 0 && 8160 - NI >= 0) ? (255) : 256);
            kernel3 <<<k3_dimGrid, k3_dimBlock>>> (dev_tmp, dev_C, dev_D, DIM2, NI, NL, NJ, NK);
            cudaCheckKernel();
          }
          
      }
    }
cudaCheckReturn(hipMemcpy(tmp, dev_tmp, (NI) * (DIM2) * sizeof(float), hipMemcpyDeviceToHost));
cudaCheckReturn(hipMemcpy(D, dev_D, (NI) * (DIM2) * sizeof(float), hipMemcpyDeviceToHost));
    
cudaCheckReturn(hipFree(dev_tmp));
cudaCheckReturn(hipFree(dev_A));
cudaCheckReturn(hipFree(dev_C));
cudaCheckReturn(hipFree(dev_B));
cudaCheckReturn(hipFree(dev_D));
  }

}




int main(int argc, char *argv[]) {

  // number of processors
  int tool_kind=1, num_proc=1;

  tool_kind = atoi(argv[1]);
  num_proc = atoi(argv[2]);

  if(argc > 3)
    N = atoi(argv[3]);

  NK = NI = NJ = NL = NM = N;
  DIM = DIM1 = DIM2 = DIM3 = N+20;

  float (*A)[DIM2], (*B)[DIM2], (*C)[DIM2], (*D)[DIM2], (*tmp)[DIM2];

  A = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  B = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  C = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  D = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  tmp = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));

  // variables to meassure time
  struct timeval s1, f1;
  double  duration, duration1;

  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  gettimeofday(&s1, NULL);

    seq(A,B,C,D,tmp);


  gettimeofday(&f1, NULL);

  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
  printf("Czas: %2.3f seconds\n", duration);


  // -----------------------------------
  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!


}









