#include "hip/hip_runtime.h"
#include "2mm_kernel.hu"
__global__ void kernel0(float *tmp, int DIM2, int NI, int NL, int NJ, int NK)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NI; g1 += 8192)
      for (int g3 = 32 * b1; g3 < NJ; g3 += 8192) {
        if (NI >= t0 + g1 + 1)
          for (int c1 = t1; c1 <= min(31, NJ - g3 - 1); c1 += 16)
            tmp[(t0 + g1) * (DIM2) + g3 + c1] = (0);
        __syncthreads();
      }
}
__global__ void kernel1(float *tmp, int alpha, float *A, float *B, int DIM2, int NI, int NL, int NJ, int NK)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    float private_tmp[1][2];
    __shared__ float shared_A[32][32];

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NI; g1 += 8192)
      for (int g3 = 32 * b1; g3 < NJ; g3 += 8192) {
        if (NJ >= t1 + g3 + 1 && NI >= t0 + g1 + 1) {
          private_tmp[0][0] = tmp[(t0 + g1) * (DIM2) + (t1 + g3)];
          if (NJ >= t1 + g3 + 17)
            private_tmp[0][1] = tmp[(t0 + g1) * (DIM2) + (t1 + g3 + 16)];
        }
        for (int g9 = 0; g9 < NK; g9 += 32) {
          if (NI >= t0 + g1 + 1 && DIM2 >= g9 + 1)
            for (int c1 = t1; c1 <= min(DIM2 - g9 - 1, 31); c1 += 16)
              shared_A[t0][c1] = A[(t0 + g1) * (DIM2) + (g9 + c1)];
          __syncthreads();
          if (NJ >= t1 + g3 + 1 && NI >= t0 + g1 + 1)
            for (int c3 = 0; c3 <= min(31, NK - g9 - 1); c3 += 1) {
              private_tmp[0][0] += ((alpha * shared_A[t0][c3]) * B[(g9 + c3) * (DIM2) + t1 + g3]);
              if (NJ >= t1 + g3 + 17)
                private_tmp[0][1] += ((alpha * shared_A[t0][c3]) * B[(g9 + c3) * (DIM2) + t1 + g3 + 16]);
            }
          __syncthreads();
        }
        if (NJ >= t1 + g3 + 1 && NI >= t0 + g1 + 1) {
          tmp[(t0 + g1) * (DIM2) + (t1 + g3)] = private_tmp[0][0];
          if (NJ >= t1 + g3 + 17)
            tmp[(t0 + g1) * (DIM2) + (t1 + g3 + 16)] = private_tmp[0][1];
        }
        __syncthreads();
      }
}
__global__ void kernel2(int beta, float *D, int DIM2, int NI, int NL, int NJ, int NK)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NI; g1 += 8192)
      for (int g3 = 32 * b1; g3 < NL; g3 += 8192) {
        if (NI >= t0 + g1 + 1)
          for (int c1 = t1; c1 <= min(31, NL - g3 - 1); c1 += 16)
            D[(t0 + g1) * (DIM2) + g3 + c1] *= beta;
        __syncthreads();
      }
}
__global__ void kernel3(float *tmp, float *C, float *D, int DIM2, int NI, int NL, int NJ, int NK)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_tmp[32][32];
    __shared__ float shared_C[32][32];
    float private_D[1][2];

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NI; g1 += 8192)
      for (int g3 = 32 * b1; g3 < NL; g3 += 8192) {
        if (NL >= t1 + g3 + 1 && NI >= t0 + g1 + 1) {
          private_D[0][0] = D[(t0 + g1) * (DIM2) + (t1 + g3)];
          if (NL >= t1 + g3 + 17)
            private_D[0][1] = D[(t0 + g1) * (DIM2) + (t1 + g3 + 16)];
        }
        for (int g9 = 0; g9 < NJ; g9 += 32) {
          if (DIM2 >= g3 + 1 && NJ >= t0 + g9 + 1)
            for (int c1 = t1; c1 <= min(31, DIM2 - g3 - 1); c1 += 16)
              shared_C[t0][c1] = C[(t0 + g9) * (DIM2) + (g3 + c1)];
          if (NI >= t0 + g1 + 1 && DIM2 >= g9 + 1)
            for (int c1 = t1; c1 <= min(DIM2 - g9 - 1, 31); c1 += 16)
              shared_tmp[t0][c1] = tmp[(t0 + g1) * (DIM2) + (g9 + c1)];
          __syncthreads();
          if (NL >= t1 + g3 + 1 && NI >= t0 + g1 + 1)
            for (int c3 = 0; c3 <= min(31, NJ - g9 - 1); c3 += 1) {
              private_D[0][0] += (shared_tmp[t0][c3] * shared_C[c3][t1]);
              if (NL >= t1 + g3 + 17)
                private_D[0][1] += (shared_tmp[t0][c3] * shared_C[c3][t1 + 16]);
            }
          __syncthreads();
        }
        if (NL >= t1 + g3 + 1 && NI >= t0 + g1 + 1) {
          D[(t0 + g1) * (DIM2) + (t1 + g3)] = private_D[0][0];
          if (NL >= t1 + g3 + 17)
            D[(t0 + g1) * (DIM2) + (t1 + g3 + 16)] = private_D[0][1];
        }
        __syncthreads();
      }
}
