#include "hip/hip_runtime.h"
#include "3mm_kernel.hu"
__global__ void kernel0(float *G, float *F, float *E, int DIM2, int NI, int NL, int NJ)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    float private_G[1][2];
    __shared__ float shared_F[32][32];

    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    #define min(x,y)    ((x) < (y) ? (x) : (y))
    for (int g1 = 32 * b0; g1 < NI; g1 += 8192)
      for (int g3 = 32 * b1; g3 < NL; g3 += 8192) {
        if (NJ >= 1)
          if (NL >= t1 + g3 + 1 && NI >= t0 + g1 + 1) {
            private_G[0][0] = G[(t0 + g1) * (DIM2) + (t1 + g3)];
            if (NL >= t1 + g3 + 17)
              private_G[0][1] = G[(t0 + g1) * (DIM2) + (t1 + g3 + 16)];
          }
        for (int g9 = 0; g9 <= (NJ >= 1 ? NJ - 1 : 0); g9 += 32) {
          if (NJ >= g9 + 1)
            if (DIM2 >= g3 + 1 && NJ >= t0 + g9 + 1)
              for (int c1 = t1; c1 <= min(31, DIM2 - g3 - 1); c1 += 16)
                shared_F[t0][c1] = F[(t0 + g9) * (DIM2) + (g3 + c1)];
          __syncthreads();
          if (g9 == 0 && NI >= t0 + g1 + 1 && NL >= t1 + g3 + 1) {
            private_G[0][0] = (0);
            if (NL >= t1 + g3 + 17)
              private_G[0][1] = (0);
          }
          if (NI >= t0 + g1 + 1 && NL >= t1 + g3 + 1)
            for (int c3 = 0; c3 <= min(31, NJ - g9 - 1); c3 += 1) {
              private_G[0][0] += (E[(t0 + g1) * (DIM2) + g9 + c3] * shared_F[c3][t1]);
              if (NL >= t1 + g3 + 17)
                private_G[0][1] += (E[(t0 + g1) * (DIM2) + g9 + c3] * shared_F[c3][t1 + 16]);
            }
          __syncthreads();
        }
        if (NL >= t1 + g3 + 1 && NI >= t0 + g1 + 1) {
          G[(t0 + g1) * (DIM2) + (t1 + g3)] = private_G[0][0];
          if (NL >= t1 + g3 + 17)
            G[(t0 + g1) * (DIM2) + (t1 + g3 + 16)] = private_G[0][1];
        }
        __syncthreads();
      }
}
