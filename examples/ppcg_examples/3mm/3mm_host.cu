#include "hip/hip_runtime.h"
#include <assert.h>
#include "3mm_kernel.hu"
#include<sys/time.h>
#include<stdlib.h>
#include<stdio.h>
#include <math.h>

#define ceild(n,d)  ceil(((double)(n))/((double)(d)))
#define floord(n,d) floor(((double)(n))/((double)(d)))
#define max(x,y)    ((x) > (y)? (x) : (y))
#define min(x,y)    ((x) < (y)? (x) : (y))


int N = 1024;
int DIM;
int NI, NJ, NL, NM, NK;
int DIM1;
int DIM2;
int DIM3;





void seq(float A[DIM1][DIM2], float B[DIM1][DIM2], float C[DIM1][DIM2], float D[DIM1][DIM2], float E[DIM1][DIM2], float F[DIM1][DIM2], float G[DIM1][DIM2]) {
int i,j,k;

    #pragma scop
	for (i=0; i<NI; i++) {
		for (j=0; j<NJ; j++) {
			E[i][j] = 0;
			for (k=0; k<NK; k++) {
				E[i][j] += A[i][k] * B[k][j];
			}
		}
	}
	#pragma endscop

    #pragma scop
	for (i=0; i<NJ; i++) {
		for (j=0; j<NL; j++) {
			F[i][j] = 0;
			for (k=0; k<NM; k++) {
				F[i][j] += C[i][k] * D[k][j];
			}
		}
	}
	#pragma endscop

if (NL >= 1 && NI >= 1)
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float *dev_G;
    float *dev_F;
    float *dev_E;
    
    cudaCheckReturn(hipMalloc((void **) &dev_G, (NI) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_F, (NJ) * (DIM2) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_E, (NI) * (DIM2) * sizeof(float)));
    
cudaCheckReturn(hipMemcpy(dev_G, G, (NI) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_F, F, (NJ) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
cudaCheckReturn(hipMemcpy(dev_E, E, (NI) * (DIM2) * sizeof(float), hipMemcpyHostToDevice));
    
    #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid((8128 - NL >= 0) ? (floord(NL + 31, 32)) : (8160 - NL >= 0 && -8129 + NL >= 0) ? (255) : 256, (8160 - NI >= 0) ? (floord(NI + 31, 32)) : 256);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_G, dev_F, dev_E, DIM2, NI, NL, NJ);
      cudaCheckKernel();
    }
    
cudaCheckReturn(hipMemcpy(G, dev_G, (NI) * (DIM2) * sizeof(float), hipMemcpyDeviceToHost));
    
cudaCheckReturn(hipFree(dev_G));
cudaCheckReturn(hipFree(dev_F));
cudaCheckReturn(hipFree(dev_E));
  }

}





int main(int argc, char *argv[]) {

  // number of processors
  int tool_kind=1, num_proc=1;

  tool_kind = atoi(argv[1]);
  num_proc = atoi(argv[2]);

  if(argc > 3)
    N = atoi(argv[3]);

  NK = NI = NJ = NL = NM = N;
  DIM = DIM1 = DIM2 = DIM3 = N+20;

  float (*A)[DIM2], (*B)[DIM2], (*C)[DIM2], (*D)[DIM2], (*E)[DIM2], (*F)[DIM2], (*G)[DIM2];

  A = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  B = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  C = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  D = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  E = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  F = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));
  G = (float (*)[DIM2])malloc(DIM1 * DIM2 * sizeof(float));

  // variables to meassure time
  struct timeval s1, f1;
  double  duration, duration1;

  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  gettimeofday(&s1, NULL);


    seq(A,B,C,D,E,F,G);

  gettimeofday(&f1, NULL);

  duration = (double)f1.tv_sec + ((double)f1.tv_usec/1000000) - ((double)s1.tv_sec + ((double)s1.tv_usec/1000000));
  printf("Czas: %2.3f seconds\n", duration);

  // -----------------------------------
  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
}

